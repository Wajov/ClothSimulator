#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <vector>
#include <cassert>


// create poisson matrix with Dirichlet bc. of a rectangular grid with
// dimension NxN
void assemble_poisson_matrix_coo(std::vector<float>& vals, std::vector<int>& row, std::vector<int>& col,
                     std::vector<float>& rhs, int Nrows, int Ncols) {

        //nnz: 5 entries per row (node) for nodes in the interior
    // 1 entry per row (node) for nodes on the boundary, since we set them explicitly to 1.
    int nnz = 5*Nrows*Ncols - (2*(Ncols-1) + 2*(Nrows-1))*4;
    vals.resize(nnz);
    row.resize(nnz);
    col.resize(nnz);
    rhs.resize(Nrows*Ncols);

    int counter = 0;
    for(int i = 0; i < Nrows; ++i) {
        for (int j = 0; j < Ncols; ++j) {
            int idx = j + Ncols*i;
            if (i == 0 || j == 0 || j == Ncols-1 || i == Nrows-1) {
                vals[counter] = 1.;
                row[counter] = idx;
                col[counter] = idx;
                counter++;
                rhs[idx] = 1.;
//                if (i == 0) {
//                    rhs[idx] = 3.;
//                }
            } else { // -laplace stencil
                // above
                vals[counter] = -1.;
                row[counter] = idx;
                col[counter] = idx-Ncols;
                counter++;
                // left
                vals[counter] = -1.;
                row[counter] = idx;
                col[counter] = idx-1;
                counter++;
                // center
                vals[counter] = 4.;
                row[counter] = idx;
                col[counter] = idx;
                counter++;
                // right
                vals[counter] = -1.;
                row[counter] = idx;
                col[counter] = idx+1;
                counter++;
                // below
                vals[counter] = -1.;
                row[counter] = idx;
                col[counter] = idx+Ncols;
                counter++;

                rhs[idx] = 0;
            }
        }
    }
    assert(counter == nnz);
}



int main() {
    // --- create library handles:
    hipsolverSpHandle_t cusolver_handle;
    hipsolverStatus_t cusolver_status;
    cusolver_status = hipsolverSpCreate(&cusolver_handle);
    std::cout << "status create cusolver handle: " << cusolver_status << std::endl;

    hipsparseHandle_t cusparse_handle;
    hipsparseStatus_t cusparse_status;
    cusparse_status = hipsparseCreate(&cusparse_handle);
    std::cout << "status create cusparse handle: " << cusparse_status << std::endl;

    // --- prepare matrix:
    int Nrows = 4;
    int Ncols = 4;
    std::vector<float> csrVal;
    std::vector<int> cooRow;
    std::vector<int> csrColInd;
    std::vector<float> b;

    assemble_poisson_matrix_coo(csrVal, cooRow, csrColInd, b, Nrows, Ncols);

    int nnz = csrVal.size();
    int m = Nrows * Ncols;
    std::vector<int> csrRowPtr(m+1);

    // --- prepare solving and copy to GPU:
    std::vector<float> x(m);
    float tol = 1e-5;
    int reorder = 0;
    int singularity = 0;

    float *db, *dcsrVal, *dx;
    int *dcsrColInd, *dcsrRowPtr, *dcooRow;
    hipMalloc((void**)&db, m*sizeof(float));
    hipMalloc((void**)&dx, m*sizeof(float));
    hipMalloc((void**)&dcsrVal, nnz*sizeof(float));
    hipMalloc((void**)&dcsrColInd, nnz*sizeof(int));
    hipMalloc((void**)&dcsrRowPtr, (m+1)*sizeof(int));
    hipMalloc((void**)&dcooRow, nnz*sizeof(int));

    hipMemcpy(db, b.data(), b.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dcsrVal, csrVal.data(), csrVal.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dcsrColInd, csrColInd.data(), csrColInd.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dcooRow, cooRow.data(), cooRow.size()*sizeof(int), hipMemcpyHostToDevice);

    cusparse_status = hipsparseXcoo2csr(cusparse_handle, dcooRow, nnz, m,
                                       dcsrRowPtr, HIPSPARSE_INDEX_BASE_ZERO);
    std::cout << "status cusparse coo2csr conversion: " << cusparse_status << std::endl;

    hipDeviceSynchronize(); // matrix format conversion has to be finished!

    // --- everything ready for computation:

    hipsparseMatDescr_t descrA;

    cusparse_status = hipsparseCreateMatDescr(&descrA);
    std::cout << "status cusparse createMatDescr: " << cusparse_status << std::endl;

    // optional: print dense matrix that has been allocated on GPU

    std::vector<float> A(m*m, 0);
    float *dA;
    hipMalloc((void**)&dA, A.size()*sizeof(float));

    hipsparseScsr2dense(cusparse_handle, m, m, descrA, dcsrVal,
                       dcsrRowPtr, dcsrColInd, dA, m);

    hipMemcpy(A.data(), dA, A.size()*sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "A: \n";
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < m; ++j) {
            std::cout << A[i*m + j] << " ";
        }
        std::cout << std::endl;
    }

    hipFree(dA);

    std::cout << "b: \n";
    hipMemcpy(b.data(), db, (m)*sizeof(int), hipMemcpyDeviceToHost);
    for (auto a : b) {
        std::cout << a << ",";
    }
    std::cout << std::endl;


    // --- solving!!!!

//    cusolver_status = hipsolverSpScsrlsvchol(cusolver_handle, m, nnz, descrA, dcsrVal,
//                       dcsrRowPtr, dcsrColInd, db, tol, reorder, dx,
//                       &singularity);

     cusolver_status = hipsolverSpScsrlsvqr(cusolver_handle, m, nnz, descrA, dcsrVal,
                        dcsrRowPtr, dcsrColInd, db, tol, reorder, dx,
                        &singularity);

    hipDeviceSynchronize();

    std::cout << "singularity (should be -1): " << singularity << std::endl;

    std::cout << "status cusolver solving (!): " << cusolver_status << std::endl;

    hipMemcpy(x.data(), dx, m*sizeof(float), hipMemcpyDeviceToHost);

    // relocated these 2 lines from above to solve (2):
    cusparse_status = hipsparseDestroy(cusparse_handle);
    std::cout << "status destroy cusparse handle: " << cusparse_status << std::endl;

    cusolver_status = hipsolverSpDestroy(cusolver_handle);
    std::cout << "status destroy cusolver handle: " << cusolver_status << std::endl;

    for (auto a : x) {
        std::cout << a << " ";
    }
    std::cout << std::endl;



    hipFree(db);
    hipFree(dx);
    hipFree(dcsrVal);
    hipFree(dcsrColInd);
    hipFree(dcsrRowPtr);
    hipFree(dcooRow);

    return 0;
}
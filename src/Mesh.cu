#include "Mesh.cuh"

Mesh::Mesh(const Json::Value &json, const Transform* transform, const Material* material) {
    load(json.asString(), transform, material);
}

Mesh::~Mesh() {
    if (!gpu) {
        for (const Node* node : nodes)
            delete node;
        for (const Vertex* vertex : vertices)
            delete vertex;
        for (const Edge* edge : edges)
            delete edge;
        for (const Face* face : faces)
            delete face;
    } else {
        deleteGpu<<<GRID_SIZE, BLOCK_SIZE>>>(nodesGpu.size(), pointer(nodesGpu));
        CUDA_CHECK_LAST();
        deleteGpu<<<GRID_SIZE, BLOCK_SIZE>>>(verticesGpu.size(), pointer(verticesGpu));
        CUDA_CHECK_LAST();
        deleteGpu<<<GRID_SIZE, BLOCK_SIZE>>>(edgesGpu.size(), pointer(edgesGpu));
        CUDA_CHECK_LAST();
        deleteGpu<<<GRID_SIZE, BLOCK_SIZE>>>(facesGpu.size(), pointer(facesGpu));
        CUDA_CHECK_LAST();
    }
}

std::vector<std::string> Mesh::split(const std::string& s, char c) const {
    std::string t = s;
    std::vector<std::string> ans;
    while (t.find(c) != std::string::npos) {
        int index = t.find(c);
        ans.push_back(t.substr(0, index));
        t.erase(0, index + 1);
    }
    ans.push_back(t);
    return ans;
}

Edge* Mesh::findEdge(int index0, int index1, std::map<Pairii, int>& edgeMap) {
    if (index0 > index1)
        mySwap(index0, index1);
    Pairii index(index0, index1);
    auto iter = edgeMap.find(index);
    if (iter != edgeMap.end())
        return edges[iter->second];
    else {
        edgeMap[index] = edges.size();
        edges.push_back(new Edge(nodes[index0], nodes[index1]));
        return edges.back();
    }
}

void Mesh::initialize(const std::vector<Vector3f>& x, const std::vector<Vector3f>& v, const std::vector<Vector2f>& u, const std::vector<int>& xIndices, const std::vector<int>& uIndices, const Material* material) {
    bool isFree = (material != nullptr);
    if (!gpu) {
        nodes.resize(x.size());
        vertices.resize(u.size());
        edges.clear();
        faces.resize(xIndices.size() / 3);
        for (int i = 0; i < x.size(); i++) {
            Node* node = new Node(x[i], isFree);
            node->v = i < v.size() ? v[i] : Vector3f(0.0f, 0.0f, 0.0f);
            nodes[i] = node;
        }
        for (int i = 0; i < u.size(); i++)
            vertices[i] = new Vertex(u[i]);

        std::map<Pairii, int> edgeMap;
        for (int i = 0; i < xIndices.size(); i += 3) {
            int xIndex0 = xIndices[i];
            int xIndex1 = xIndices[i + 1];
            int xIndex2 = xIndices[i + 2];
            int uIndex0 = uIndices[i];
            int uIndex1 = uIndices[i + 1];
            int uIndex2 = uIndices[i + 2];

            Vertex* vertex0 = vertices[uIndex0];
            Vertex* vertex1 = vertices[uIndex1];
            Vertex* vertex2 = vertices[uIndex2];
            
            Edge* edge0 = findEdge(xIndex0, xIndex1, edgeMap);
            Edge* edge1 = findEdge(xIndex1, xIndex2, edgeMap);
            Edge* edge2 = findEdge(xIndex2, xIndex0, edgeMap);
            Face* face = new Face(vertex0, vertex1, vertex2, material);

            vertex0->node = nodes[xIndex0];
            vertex1->node = nodes[xIndex1];
            vertex2->node = nodes[xIndex2];
            edge0->initialize(vertex2, face);
            edge1->initialize(vertex0, face);
            edge2->initialize(vertex1, face);
            face->setEdges(edge0, edge1, edge2);

            faces[i / 3] = face;
        }

        for (const Edge* edge : edges)
            if (edge->isBoundary() || edge->isSeam())
                for (int i = 0; i < 2; i++)
                    edge->nodes[i]->preserve = true;
    } else {
        int nNodes = x.size();
        int nVertices = u.size();
        int nFaces = xIndices.size() / 3;
        int nEdges = xIndices.size();
        thrust::device_vector<Vector3f> xGpu = x;
        thrust::device_vector<Vector3f> vGpu = v;
        thrust::device_vector<Vector2f> uGpu = u;
        thrust::device_vector<int> xIndicesGpu = xIndices;
        thrust::device_vector<int> uIndicesGpu = uIndices;

        nodesGpu.resize(nNodes);
        Node** nodesPointer = pointer(nodesGpu);
        initializeNodes<<<GRID_SIZE, BLOCK_SIZE>>>(nNodes, pointer(xGpu), isFree, v.size(), pointer(vGpu), nodesPointer);
        CUDA_CHECK_LAST();

        verticesGpu.resize(nVertices);
        Vertex** verticesPointer = pointer(verticesGpu);
        initializeVertices<<<GRID_SIZE, BLOCK_SIZE>>>(nVertices, pointer(uGpu), verticesPointer);
        CUDA_CHECK_LAST();

        facesGpu.resize(nFaces);
        Face** facesPointer = pointer(facesGpu);
        thrust::device_vector<Pairii> edgeIndices(nEdges);
        Pairii* edgeIndicesPointer = pointer(edgeIndices);
        thrust::device_vector<EdgeData> edgeData(nEdges);
        EdgeData* edgeDataPointer = pointer(edgeData);
        initializeFaces<<<GRID_SIZE, BLOCK_SIZE>>>(nFaces, pointer(xIndicesGpu), pointer(uIndicesGpu), nodesPointer, material, verticesPointer, facesPointer, edgeIndicesPointer, edgeDataPointer);
        CUDA_CHECK_LAST();
        thrust::sort_by_key(edgeIndices.begin(), edgeIndices.end(), edgeData.begin());

        edgesGpu.resize(nEdges);
        Edge** edgesPointer = pointer(edgesGpu);
        initializeEdges<<<GRID_SIZE, BLOCK_SIZE>>>(nEdges, edgeIndicesPointer, edgeDataPointer, nodesPointer, edgesPointer);
        CUDA_CHECK_LAST();
        setEdges<<<GRID_SIZE, BLOCK_SIZE>>>(nEdges, edgeIndicesPointer, edgeDataPointer, edgesPointer);
        CUDA_CHECK_LAST();
        edgesGpu.erase(thrust::remove(edgesGpu.begin(), edgesGpu.end(), nullptr), edgesGpu.end());

        nEdges = edgesGpu.size();
        setPreserve<<<GRID_SIZE, BLOCK_SIZE>>>(nEdges, edgesPointer);
        CUDA_CHECK_LAST();
    }

    updateIndices();
    updateStructures();
    updateNodeGeometries();
    updateFaceGeometries();
}

std::vector<Node*>& Mesh::getNodes() {
    return nodes;
}

thrust::device_vector<Node*>& Mesh::getNodesGpu() {
    return nodesGpu;
}

std::vector<Vertex*>& Mesh::getVertices() {
    return vertices;
}

thrust::device_vector<Vertex*>& Mesh::getVerticesGpu() {
    return verticesGpu;
}

std::vector<Edge*>& Mesh::getEdges() {
    return edges;
}

thrust::device_vector<Edge*>& Mesh::getEdgesGpu() {
    return edgesGpu;
}

std::vector<Face*>& Mesh::getFaces() {
    return faces;
}

thrust::device_vector<Face*>& Mesh::getFacesGpu() {
    return facesGpu;
}

bool Mesh::contain(const Vertex* vertex) const {
    int index = vertex->index;
    return index < vertices.size() && vertices[index] == vertex;
}

bool Mesh::contain(const Face* face) const {
    return contain(face->vertices[0]) && contain(face->vertices[1]) && contain(face->vertices[2]);
}

void Mesh::reset() {
    if (!gpu)
        for (Node* node : nodes)
            node->x = node->x0;
    else {
        resetGpu<<<GRID_SIZE, BLOCK_SIZE>>>(nodesGpu.size(), pointer(nodesGpu));
        CUDA_CHECK_LAST();
    }
}

std::vector<BackupFace> Mesh::backupFaces() const {
    std::vector<BackupFace> ans(faces.size());
    for (int i = 0; i < faces.size(); i++) {
        BackupFace& backupFace = ans[i];
        Face* face = faces[i];
        for (int j = 0; j < 3; j++) {
            Vertex* vertex = face->vertices[j];
            backupFace.x[j] = vertex->node->x;
            backupFace.u[j] = vertex->u;
        }
    }
    return ans;
}

thrust::device_vector<BackupFace> Mesh::backupFacesGpu() const {
    thrust::device_vector<BackupFace> ans(facesGpu.size());
    setBackupFaces<<<GRID_SIZE, BLOCK_SIZE>>>(facesGpu.size(), pointer(facesGpu), pointer(ans));
    CUDA_CHECK_LAST();

    return ans;
}

void Mesh::apply(const Operator& op) {
    if (!gpu) {
        for (const Node* node : op.removedNodes)
            nodes.erase(std::remove(nodes.begin(), nodes.end(), node), nodes.end());
        nodes.insert(nodes.end(), op.addedNodes.begin(), op.addedNodes.end());

        for (const Vertex* vertex : op.removedVertices)
            vertices.erase(std::remove(vertices.begin(), vertices.end(), vertex), vertices.end());
        vertices.insert(vertices.end(), op.addedVertices.begin(), op.addedVertices.end());
        
        for (const Edge* edge : op.removedEdges)
            edges.erase(std::remove(edges.begin(), edges.end(), edge), edges.end());
        edges.insert(edges.end(), op.addedEdges.begin(), op.addedEdges.end());

        for (const Face* face : op.removedFaces)
            faces.erase(std::remove(faces.begin(), faces.end(), face), faces.end());
        faces.insert(faces.end(), op.addedFaces.begin(), op.addedFaces.end());

        for (const Node* node : op.removedNodes)
            delete node;
        for (const Vertex* vertex : op.removedVertices)
            delete vertex;
        for (const Edge* edge : op.removedEdges)
            delete edge;
        for (const Face* face : op.removedFaces)
            delete face;
    } else {
        removeGpu(op.removedNodesGpu, nodesGpu);
        nodesGpu.insert(nodesGpu.end(), op.addedNodesGpu.begin(), op.addedNodesGpu.end());
        removeGpu(op.removedVerticesGpu, verticesGpu);
        verticesGpu.insert(verticesGpu.end(), op.addedVerticesGpu.begin(), op.addedVerticesGpu.end());
        removeGpu(op.removedEdgesGpu, edgesGpu);
        edgesGpu.insert(edgesGpu.end(), op.addedEdgesGpu.begin(), op.addedEdgesGpu.end());
        removeGpu(op.removedFacesGpu, facesGpu);
        facesGpu.insert(facesGpu.end(), op.addedFacesGpu.begin(), op.addedFacesGpu.end());

        deleteGpu<<<GRID_SIZE, BLOCK_SIZE>>>(op.removedNodesGpu.size(), pointer(op.removedNodesGpu));
        CUDA_CHECK_LAST();
        deleteGpu<<<GRID_SIZE, BLOCK_SIZE>>>(op.removedVerticesGpu.size(), pointer(op.removedVerticesGpu));
        CUDA_CHECK_LAST();
        deleteGpu<<<GRID_SIZE, BLOCK_SIZE>>>(op.removedEdgesGpu.size(), pointer(op.removedEdgesGpu));
        CUDA_CHECK_LAST();
        deleteGpu<<<GRID_SIZE, BLOCK_SIZE>>>(op.removedFacesGpu.size(), pointer(op.removedFacesGpu));
        CUDA_CHECK_LAST();
    }
}

void Mesh::updateIndices() {
    if (!gpu) {
        for (int i = 0; i < nodes.size(); i++)
            nodes[i]->index = i;
        for (int i = 0; i < vertices.size(); i++)
            vertices[i]->index = i;
    } else {
        updateNodeIndices<<<GRID_SIZE, BLOCK_SIZE>>>(nodesGpu.size(), pointer(nodesGpu));
        CUDA_CHECK_LAST();

        updateVertexIndices<<<GRID_SIZE, BLOCK_SIZE>>>(verticesGpu.size(), pointer(verticesGpu));
        CUDA_CHECK_LAST();
    }
}

void Mesh::updateStructures() {
    if (!gpu) {
        for (Node* node : nodes) {
            node->mass = 0.0f;
            node->area = 0.0f;
        }        
        for (const Face* face : faces) {
            float mass = face->mass / 3.0f;
            float area = face->area;
            for (int i = 0; i < 3; i++) {
                Node* node = face->vertices[i]->node;
                node->mass += mass;
                node->area += area;
            }
        }
    } else {
        initializeNodeStructures<<<GRID_SIZE, BLOCK_SIZE>>>(nodesGpu.size(), pointer(nodesGpu));
        CUDA_CHECK_LAST();

        updateNodeStructures<<<GRID_SIZE, BLOCK_SIZE>>>(facesGpu.size(), pointer(facesGpu));
        CUDA_CHECK_LAST();
    }
}

void Mesh::updateNodeGeometries() {
    if (!gpu) {
        for (Node* node : nodes) {
            node->x1 = node->x;
            node->n = Vector3f();
        }
        for (const Face* face : faces)
            for (int i = 0; i < 3; i++) {
                Node* node = face->vertices[i]->node;
                Vector3f e0 = face->vertices[(i + 1) % 3]->node->x - node->x;
                Vector3f e1 = face->vertices[(i + 2) % 3]->node->x - node->x;
                node->n += e0.cross(e1) / (e0.norm2() * e1.norm2());
            }
        for (Node* node : nodes)
            node->n.normalize();
    } else {
        initializeNodeGeometries<<<GRID_SIZE, BLOCK_SIZE>>>(nodesGpu.size(), pointer(nodesGpu));
        CUDA_CHECK_LAST();

        updateNodeGeometriesGpu<<<GRID_SIZE, BLOCK_SIZE>>>(facesGpu.size(), pointer(facesGpu));
        CUDA_CHECK_LAST();

        finalizeNodeGeometries<<<GRID_SIZE, BLOCK_SIZE>>>(nodesGpu.size(), pointer(nodesGpu));
        CUDA_CHECK_LAST();
    }
}

void Mesh::updateFaceGeometries() {
    if (!gpu)
        for (Face* face : faces)
            face->update();
    else {
        updateFaceGeometriesGpu<<<GRID_SIZE, BLOCK_SIZE>>>(facesGpu.size(), pointer(facesGpu));
        CUDA_CHECK_LAST();
    }
}

void Mesh::updateVelocities(float dt) {
    float invDt = 1.0f / dt;
    if (!gpu)
        for (Node* node : nodes)
            node->v = (node->x - node->x0) * invDt;
    else {
        updateVelocitiesGpu<<<GRID_SIZE, BLOCK_SIZE>>>(nodesGpu.size(), invDt, pointer(nodesGpu));
        CUDA_CHECK_LAST();
    }
}

void Mesh::updateRenderingData(bool rebind) {
    if (!gpu) {
        std::vector<Renderable> renderables(3 * faces.size());
        for (int i = 0; i < faces.size(); i++) {
            Face* face = faces[i];
            for (int j = 0; j < 3; j++) {
                Vertex* vertex = face->vertices[j];
                Node* node = vertex->node;
                int index = 3 * i + j;
                renderables[index].x = node->x;
                renderables[index].n = node->n;
                renderables[index].u = vertex->u;
            }
        }

        glBindBuffer(GL_ARRAY_BUFFER, vbo);
        glBufferData(GL_ARRAY_BUFFER, renderables.size() * sizeof(Renderable), renderables.data(), GL_DYNAMIC_DRAW);
    } else {
        if (rebind) {
            glBindBuffer(GL_ARRAY_BUFFER, vbo);
            glBufferData(GL_ARRAY_BUFFER, 3 * facesGpu.size() * sizeof(Renderable), nullptr, GL_DYNAMIC_DRAW);

            CUDA_CHECK(hipGraphicsGLRegisterBuffer(&vboCuda, vbo, hipGraphicsRegisterFlagsWriteDiscard));
        }

        CUDA_CHECK(hipGraphicsMapResources(1, &vboCuda));
        Renderable* renderables;
        size_t nRenderanles;
        CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&renderables), &nRenderanles, vboCuda));
        updateRenderingDataGpu<<<GRID_SIZE, BLOCK_SIZE>>>(facesGpu.size(), pointer(facesGpu), renderables);
        CUDA_CHECK_LAST();

        CUDA_CHECK(hipGraphicsUnmapResources(1, &vboCuda));
    }
}

void Mesh::bind() {
    glGenVertexArrays(1, &vao);
    glBindVertexArray(vao);
    glGenBuffers(1, &vbo);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(Renderable), reinterpret_cast<void*>(offsetof(Renderable, x)));
    glEnableVertexAttribArray(1);
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, sizeof(Renderable), reinterpret_cast<void*>(offsetof(Renderable, n)));
    glEnableVertexAttribArray(2);
    glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, sizeof(Renderable), reinterpret_cast<void*>(offsetof(Renderable, u)));

    updateRenderingData(true);
}

void Mesh::render() const {
    glBindVertexArray(vao);
    glDrawArrays(GL_TRIANGLES, 0, 3 * (!gpu ? faces.size() : facesGpu.size()));
    glBindVertexArray(0);
}

void Mesh::load(const std::string& path, const Transform* transform, const Material* material) {
    std::ifstream fin(path);
    if (!fin.is_open()) {
        std::cerr << "Failed to open mesh file: " << path << std::endl;
        exit(1);
    }

    std::string line;
    std::vector<Vector3f> x, v;
    std::vector<Vector2f> u;
    std::vector<int> xIndices, uIndices;
    while (getline(fin, line)) {
        std::vector<std::string> s = std::move(split(line, ' '));
        if (s[0] == "v")
            x.push_back(transform->applyToPoint(Vector3f(std::stod(s[1]), std::stod(s[2]), std::stod(s[3]))));
        else if (s[0] == "nv")
            v.push_back(transform->applyToVector(Vector3f(std::stod(s[1]), std::stod(s[2]), std::stod(s[3]))));
        else if (s[0] == "vt")
            u.emplace_back(std::stof(s[1]), std::stof(s[2]));
        else if (s[0] == "f")
            for (int i = 1; i < 4; i++)
                if (line.find('/') != std::string::npos) {
                    std::vector<std::string> t = std::move(split(s[i], '/'));
                    xIndices.push_back(std::stoi(t[0]) - 1);
                    uIndices.push_back(std::stoi(t[1]) - 1);
                } else {
                    u.emplace_back(0.0f, 0.0f);
                    xIndices.push_back(std::stoi(s[i]) - 1);
                    uIndices.push_back(u.size() - 1);
                }
    }
    fin.close();

    initialize(x, v, u, xIndices, uIndices, material);
}

void Mesh::save(const std::string& path) {
    std::ofstream fout(path);
    if (!gpu) {
        for (const Node* node : nodes) {
            fout << "v " << node->x(0) << " " << node->x(1) << " " << node->x(2) << std::endl;
            fout << "nv " << node->v(0) << " " << node->v(1) << " " << node->v(2) << std::endl;
        }
        for (const Vertex* vertex : vertices)
            fout << "vt " << vertex->u(0) << " " << vertex->u(1) << std::endl;
        for (const Face* face : faces) {
            fout << "f";
            for (int i = 0; i < 3; i++) {
                Vertex* vertex = face->vertices[i];
                int xIndex = vertex->node->index + 1;
                int uIndex = vertex->index + 1;
                fout << " " << xIndex << "/" << uIndex;
            }
            fout << std::endl;
        }
    } else {
        int nNodes = nodesGpu.size();
        thrust::device_vector<Vector3f> x(nNodes);
        copyX<<<GRID_SIZE, BLOCK_SIZE>>>(nNodes, pointer(nodesGpu), pointer(x));
        CUDA_CHECK_LAST();
        for (const Vector3f& xt : x)
            fout << "v " << xt(0) << " " << xt(1) << " " << xt(2) << std::endl;

        thrust::device_vector<Vector3f> v(nNodes);
        copyV<<<GRID_SIZE, BLOCK_SIZE>>>(nNodes, pointer(nodesGpu), pointer(v));
        CUDA_CHECK_LAST();
        for (const Vector3f& vt : v)
            fout << "nv " << vt(0) << " " << vt(1) << " " << vt(2) << std::endl;

        int nVertices = verticesGpu.size();
        thrust::device_vector<Vector2f> u(nVertices);
        copyU<<<GRID_SIZE, BLOCK_SIZE>>>(nVertices, pointer(verticesGpu), pointer(u));
        CUDA_CHECK_LAST();
        for (const Vector2f& ut : u)
            fout << "vt " << ut(0) << " " << ut(1) << std::endl;
        
        int nFaces = facesGpu.size();
        thrust::device_vector<Pairii> indices(3 * nFaces);
        copyFaceIndices<<<GRID_SIZE, BLOCK_SIZE>>>(nFaces, pointer(facesGpu), pointer(indices));
        CUDA_CHECK_LAST();
        for (int i = 0; i < nFaces; i++) {
            fout << "f";
            for (int j = 0; j < 3; j++) {
                const Pairii& index = indices[3 * i + j];
                fout << " " << index.first + 1 << "/" << index.second + 1;
            }
            fout << std::endl;
        }

        fout.close();
    }
}

void Mesh::check() const {
    if (!gpu) {
        for (const Edge* edge : edges)
            for (int i = 0; i < 2; i++)
                if (edge->opposites[i] != nullptr) {
                    if (edge->vertices[i][0]->node != edge->nodes[0] || edge->vertices[i][1]->node != edge->nodes[1])
                        std::cerr << "Edge vertices check error!" << std::endl;
                    if (edge->adjacents[i] == nullptr || !edge->adjacents[i]->contain(edge->opposites[i]) || !edge->adjacents[i]->contain(edge))
                        std::cerr << "Edge adjacents check error!" << std::endl;
                } else if (edge->adjacents[i] != nullptr)
                    std::cerr << "Edge opposites check error!" << std::endl;
            
        for (const Face* face : faces)
            for (int i = 0; i < 3; i++) {
                Edge* edge = face->edges[i];
                if (edge->adjacents[0] != face && edge->adjacents[1] != face)
                    std::cerr << "Face edges check error!" << std::endl;
            }
    } else {
        checkEdges<<<GRID_SIZE, BLOCK_SIZE>>>(edgesGpu.size(), pointer(edgesGpu));
        CUDA_CHECK_LAST();

        checkFaces<<<GRID_SIZE, BLOCK_SIZE>>>(facesGpu.size(), pointer(facesGpu));
        CUDA_CHECK_LAST();
    }
}
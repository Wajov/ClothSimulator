#include "Mesh.cuh"

Mesh::Mesh(const Json::Value &json, const Transform* transform, const Material* material) {
    std::ifstream fin(json.asString());
    if (!fin.is_open()) {
        std::cerr << "Failed to open mesh file: " << json.asString() << std::endl;
        exit(1);
    }

    std::string line;
    std::vector<Vector3f> x;
    std::vector<Vector2f> u;
    std::vector<int> xIndices, uIndices;
    while (getline(fin, line)) {
        std::vector<std::string> s = std::move(split(line, ' '));
        if (s[0] == "v")
            x.push_back(transform->applyTo(Vector3f(std::stod(s[1]), std::stod(s[2]), std::stod(s[3]))));
        else if (s[0] == "vt")
            u.emplace_back(std::stof(s[1]), std::stof(s[2]));
        else if (s[0] == "f")
            for (int i = 1; i < 4; i++)
                if (line.find('/') != std::string::npos) {
                    std::vector<std::string> t = std::move(split(s[i], '/'));
                    xIndices.push_back(std::stoi(t[0]) - 1);
                    uIndices.push_back(std::stoi(t[1]) - 1);
                } else {
                    u.emplace_back(0.0f, 0.0f);
                    xIndices.push_back(std::stoi(s[i]) - 1);
                    uIndices.push_back(u.size() - 1);
                }
    }
    fin.close();

    initialize(x, u, xIndices, uIndices, material);
}

Mesh::Mesh(const Mesh* mesh) {
    nodes.resize(mesh->nodes.size());
    for (int i = 0; i < mesh->nodes.size(); i++) {
        Node* node = mesh->nodes[i];
        nodes[i] = new Node(node->x, node->isFree);
    }

    vertices.resize(mesh->vertices.size());
    for (int i = 0; i < mesh->vertices.size(); i++) {
        Vertex* vertex = mesh->vertices[i];
        vertices[i] = new Vertex(vertex->u);
        vertices[i]->node = nodes[vertex->node->index];
    }
    
    edges.resize(mesh->edges.size());
    for (int i = 0; i < mesh->edges.size(); i++) {
        Edge* edge = mesh->edges[i];
        edges[i] = new Edge(nodes[edge->nodes[0]->index], nodes[edge->nodes[1]->index]);
    }
    
    faces.resize(mesh->faces.size());
    for (int i = 0; i < mesh->faces.size(); i++) {
        Face* face = mesh->faces[i];
        faces[i] = new Face(vertices[face->vertices[0]->index], vertices[face->vertices[1]->index], vertices[face->vertices[2]->index], nullptr);
    }
}

Mesh::~Mesh() {
    if (!gpu) {
        for (const Node* node : nodes)
            delete node;
        for (const Vertex* vertex : vertices)
            delete vertex;
        for (const Edge* edge : edges)
            delete edge;
        for (const Face* face : faces)
            delete face;
    } else {
        deleteNodes<<<GRID_SIZE, BLOCK_SIZE>>>(nodesGpu.size(), pointer(nodesGpu));
        CUDA_CHECK_LAST();
        deleteVertices<<<GRID_SIZE, BLOCK_SIZE>>>(verticesGpu.size(), pointer(verticesGpu));
        CUDA_CHECK_LAST();
        deleteEdges<<<GRID_SIZE, BLOCK_SIZE>>>(edgesGpu.size(), pointer(edgesGpu));
        CUDA_CHECK_LAST();
        deleteFaces<<<GRID_SIZE, BLOCK_SIZE>>>(facesGpu.size(), pointer(facesGpu));
        CUDA_CHECK_LAST();
    }
}

std::vector<std::string> Mesh::split(const std::string& s, char c) const {
    std::string t = s;
    std::vector<std::string> ans;
    while (t.find(c) != std::string::npos) {
        int index = t.find(c);
        ans.push_back(t.substr(0, index));
        t.erase(0, index + 1);
    }
    ans.push_back(t);
    return ans;
}

Edge* Mesh::findEdge(int index0, int index1, std::map<Pairii, int>& edgeMap) {
    if (index0 > index1)
        mySwap(index0, index1);
    Pairii index(index0, index1);
    auto iter = edgeMap.find(index);
    if (iter != edgeMap.end())
        return edges[iter->second];
    else {
        edgeMap[index] = edges.size();
        edges.push_back(new Edge(nodes[index0], nodes[index1]));
        return edges.back();
    }
}

void Mesh::initialize(const std::vector<Vector3f>& x, const std::vector<Vector2f>& u, const std::vector<int>& xIndices, const std::vector<int>& uIndices, const Material* material) {
    bool isFree = (material != nullptr);
    if (!gpu) {
        nodes.resize(x.size());
        vertices.resize(u.size());
        faces.resize(xIndices.size() / 3);
        for (int i = 0; i < x.size(); i++)
            nodes[i] = new Node(x[i], isFree);
        for (int i = 0; i < u.size(); i++)
            vertices[i] = new Vertex(u[i]);

        std::map<Pairii, int> edgeMap;
        for (int i = 0; i < xIndices.size(); i += 3) {
            int xIndex0 = xIndices[i];
            int xIndex1 = xIndices[i + 1];
            int xIndex2 = xIndices[i + 2];
            int uIndex0 = uIndices[i];
            int uIndex1 = uIndices[i + 1];
            int uIndex2 = uIndices[i + 2];

            Vertex* vertex0 = vertices[uIndex0];
            Vertex* vertex1 = vertices[uIndex1];
            Vertex* vertex2 = vertices[uIndex2];
            
            Edge* edge0 = findEdge(xIndex0, xIndex1, edgeMap);
            Edge* edge1 = findEdge(xIndex1, xIndex2, edgeMap);
            Edge* edge2 = findEdge(xIndex2, xIndex0, edgeMap);
            Face* face = new Face(vertex0, vertex1, vertex2, material);

            vertex0->node = nodes[xIndex0];
            vertex1->node = nodes[xIndex1];
            vertex2->node = nodes[xIndex2];
            edge0->initialize(vertex2, face);
            edge1->initialize(vertex0, face);
            edge2->initialize(vertex1, face);
            face->setEdges(edge0, edge1, edge2);

            faces[i / 3] = face;
        }

        for (const Edge* edge : edges)
            if (edge->isBoundary() || edge->isSeam())
                for (int i = 0; i < 2; i++)
                    edge->nodes[i]->preserve = true;
    } else {
        int nNodes = x.size();
        int nVertices = u.size();
        int nFaces = xIndices.size() / 3;
        int nEdges = xIndices.size();
        thrust::device_vector<Vector3f> xGpu = x;
        thrust::device_vector<Vector2f> uGpu = u;
        thrust::device_vector<int> xIndicesGpu = xIndices;
        thrust::device_vector<int> uIndicesGpu = uIndices;

        nodesGpu.resize(nNodes);
        Node** nodesPointer = pointer(nodesGpu);
        initializeNodes<<<GRID_SIZE, BLOCK_SIZE>>>(nNodes, pointer(xGpu), isFree, nodesPointer);
        CUDA_CHECK_LAST();

        verticesGpu.resize(nVertices);
        Vertex** verticesPointer = pointer(verticesGpu);
        initializeVertices<<<GRID_SIZE, BLOCK_SIZE>>>(nVertices, pointer(uGpu), verticesPointer);
        CUDA_CHECK_LAST();

        facesGpu.resize(nFaces);
        Face** facesPointer = pointer(facesGpu);
        thrust::device_vector<Pairii> edgeIndices(nEdges);
        Pairii* edgeIndicesPointer = pointer(edgeIndices);
        thrust::device_vector<EdgeData> edgeData(nEdges);
        EdgeData* edgeDataPointer = pointer(edgeData);
        initializeFaces<<<GRID_SIZE, BLOCK_SIZE>>>(nFaces, pointer(xIndicesGpu), pointer(uIndicesGpu), nodesPointer, material, verticesPointer, facesPointer, edgeIndicesPointer, edgeDataPointer);
        CUDA_CHECK_LAST();
        thrust::sort_by_key(edgeIndices.begin(), edgeIndices.end(), edgeData.begin());

        edgesGpu.resize(nEdges);
        Edge** edgesPointer = pointer(edgesGpu);
        initializeEdges<<<GRID_SIZE, BLOCK_SIZE>>>(nEdges, edgeIndicesPointer, edgeDataPointer, nodesPointer, edgesPointer);
        CUDA_CHECK_LAST();
        setEdges<<<GRID_SIZE, BLOCK_SIZE>>>(nEdges, edgeIndicesPointer, edgeDataPointer, edgesPointer);
        CUDA_CHECK_LAST();
        edgesGpu.erase(thrust::remove_if(edgesGpu.begin(), edgesGpu.end(), IsNull()), edgesGpu.end());

        nEdges = edgesGpu.size();
        thrust::device_vector<int> nodeIndices(2 * nEdges);
        int* nodeIndicesPointer = pointer(nodeIndices);
        collectPreservedNodes<<<GRID_SIZE, BLOCK_SIZE>>>(nEdges, edgesPointer, nodeIndicesPointer);
        CUDA_CHECK_LAST();
        thrust::sort(nodeIndices.begin(), nodeIndices.end());
        auto iter = thrust::unique(nodeIndices.begin(), nodeIndices.end());
        setPreservedNodes<<<GRID_SIZE, BLOCK_SIZE>>>(iter - nodeIndices.begin(), nodeIndicesPointer, nodesPointer);
        CUDA_CHECK_LAST();
    }

    updateStructures();
    updateNodeGeometries();
    updateFaceGeometries();
}

std::vector<Node*>& Mesh::getNodes() {
    return nodes;
}

thrust::device_vector<Node*>& Mesh::getNodesGpu() {
    return nodesGpu;
}

std::vector<Vertex*>& Mesh::getVertices() {
    return vertices;
}

thrust::device_vector<Vertex*>& Mesh::getVerticesGpu() {
    return verticesGpu;
}

std::vector<Edge*>& Mesh::getEdges() {
    return edges;
}

thrust::device_vector<Edge*>& Mesh::getEdgesGpu() {
    return edgesGpu;
}

std::vector<Face*>& Mesh::getFaces() {
    return faces;
}

thrust::device_vector<Face*>& Mesh::getFacesGpu() {
    return facesGpu;
}

bool Mesh::contain(const Vertex* vertex) const {
    int index = vertex->index;
    return index < vertices.size() && vertices[index] == vertex;
}

bool Mesh::contain(const Face* face) const {
    return contain(face->vertices[0]) && contain(face->vertices[1]) && contain(face->vertices[2]);
}

void Mesh::reset() {
    if (!gpu)
        for (Node* node : nodes)
            node->x = node->x0;
    else {
        resetGpu<<<GRID_SIZE, BLOCK_SIZE>>>(nodesGpu.size(), pointer(nodesGpu));
        CUDA_CHECK_LAST();
    }
}

Vector3f Mesh::oldPosition(const Vector2f& u) const {
    for (const Face* face : faces) {
        Vector3f b = face->barycentricCoordinates(u);
        if (b(0) >= -1e-6f && b(1) >= -1e-6f && b(2) >= -1e-5f)
            return face->position(b);
    }
}

void Mesh::apply(const Operator& op) {
    for (const Node* node : op.removedNodes)
        nodes.erase(std::remove(nodes.begin(), nodes.end(), node), nodes.end());
    nodes.insert(nodes.end(), op.addedNodes.begin(), op.addedNodes.end());

    for (const Vertex* vertex : op.removedVertices)
        vertices.erase(std::remove(vertices.begin(), vertices.end(), vertex), vertices.end());
    vertices.insert(vertices.end(), op.addedVertices.begin(), op.addedVertices.end());
    
    for (const Edge* edge : op.removedEdges)
        edges.erase(std::remove(edges.begin(), edges.end(), edge), edges.end());
    edges.insert(edges.end(), op.addedEdges.begin(), op.addedEdges.end());

    for (const Face* face : op.removedFaces)
        faces.erase(std::remove(faces.begin(), faces.end(), face), faces.end());
    faces.insert(faces.end(), op.addedFaces.begin(), op.addedFaces.end());

    for (const Node* node : op.removedNodes)
        delete node;
    for (const Vertex* vertex : op.removedVertices)
        delete vertex;
    for (const Edge* edge : op.removedEdges)
        delete edge;
    for (const Face* face : op.removedFaces)
        delete face;
}

void Mesh::updateStructures() {
    if (!gpu) {
        for (int i = 0; i < nodes.size(); i++) {
            Node* node = nodes[i];
            node->index = i;
            node->mass = 0.0f;
            node->area = 0.0f;
        }
        for (int i = 0; i < vertices.size(); i++)
            vertices[i]->index = i;
        for (const Face* face : faces) {
            float mass = face->mass / 3.0f;
            float area = face->area;
            for (int i = 0; i < 3; i++) {
                Node* node = face->vertices[i]->node;
                node->mass += mass;
                node->area += area;
            }
        }
    } else {
        updateNodeIndices<<<GRID_SIZE, BLOCK_SIZE>>>(nodesGpu.size(), pointer(nodesGpu));
        CUDA_CHECK_LAST();
        updateVertexIndices<<<GRID_SIZE, BLOCK_SIZE>>>(verticesGpu.size(), pointer(verticesGpu));
        CUDA_CHECK_LAST();

        thrust::device_vector<int> indices(3 * facesGpu.size());
        thrust::device_vector<NodeData> nodeData(3 * facesGpu.size());
        collectNodeStructures<<<GRID_SIZE, BLOCK_SIZE>>>(facesGpu.size(), pointer(facesGpu), pointer(indices), pointer(nodeData));
        CUDA_CHECK_LAST();
        
        thrust::sort_by_key(indices.begin(), indices.end(), nodeData.begin());
        thrust::device_vector<int> outputIndices(3 * facesGpu.size());
        thrust::device_vector<NodeData> outputNodeData(3 * facesGpu.size());
        auto iter = thrust::reduce_by_key(indices.begin(), indices.end(), nodeData.begin(), outputIndices.begin(), outputNodeData.begin());
        setNodeStructures<<<GRID_SIZE, BLOCK_SIZE>>>(iter.first - outputIndices.begin(), pointer(outputIndices), pointer(outputNodeData), pointer(nodesGpu));
        CUDA_CHECK_LAST();
    }
}

void Mesh::updateNodeGeometries() {
    if (!gpu) {
        for (Node* node : nodes) {
            node->x1 = node->x;
            node->n = Vector3f();
        }
        for (const Face* face : faces)
            for (int i = 0; i < 3; i++) {
                Node* node = face->vertices[i]->node;
                Vector3f e0 = face->vertices[(i + 1) % 3]->node->x - node->x;
                Vector3f e1 = face->vertices[(i + 2) % 3]->node->x - node->x;
                node->n += e0.cross(e1) / (e0.norm2() * e1.norm2());
            }
        for (Node* node : nodes)
            node->n.normalize();
    } else {
        updateNodeGeometriesGpu<<<GRID_SIZE, BLOCK_SIZE>>>(nodesGpu.size(), pointer(nodesGpu));
        CUDA_CHECK_LAST();

        thrust::device_vector<int> indices(3 * facesGpu.size());
        thrust::device_vector<Vector3f> nodeData(3 * facesGpu.size());
        collectNodeGeometries<<<GRID_SIZE, BLOCK_SIZE>>>(facesGpu.size(), pointer(facesGpu), pointer(indices), pointer(nodeData));
        CUDA_CHECK_LAST();

        thrust::sort_by_key(indices.begin(), indices.end(), nodeData.begin());
        thrust::device_vector<int> outputIndices(3 * facesGpu.size());
        thrust::device_vector<Vector3f> outputNodeData(3 * facesGpu.size());
        auto iter = thrust::reduce_by_key(indices.begin(), indices.end(), nodeData.begin(), outputIndices.begin(), outputNodeData.begin());
        setNodeGeometries<<<GRID_SIZE, BLOCK_SIZE>>>(iter.first - outputIndices.begin(), pointer(outputIndices), pointer(outputNodeData), pointer(nodesGpu));
        CUDA_CHECK_LAST();
    }
}

void Mesh::updateFaceGeometries() {
    if (!gpu)
        for (Face* face : faces)
            face->update();
    else {
        updateFaceGeometriesGpu<<<GRID_SIZE, BLOCK_SIZE>>>(facesGpu.size(), pointer(facesGpu));
        CUDA_CHECK_LAST();
    }
}

void Mesh::updateVelocities(float dt) {
    float invDt = 1.0f / dt;
    if (!gpu)
        for (Node* node : nodes)
            node->v = (node->x - node->x0) * invDt;
    else {
        updateVelocitiesGpu<<<GRID_SIZE, BLOCK_SIZE>>>(nodesGpu.size(), invDt, pointer(nodesGpu));
        CUDA_CHECK_LAST();
    }
}

void Mesh::updateRenderingData(bool rebind) {
    if (!gpu) {
        std::vector<Renderable> renderables(3 * faces.size());
        for (int i = 0; i < faces.size(); i++) {
            Face* face = faces[i];
            for (int j = 0; j < 3; j++) {
                Vertex* vertex = face->vertices[j];
                Node* node = vertex->node;
                int index = 3 * i + j;
                renderables[index].x = node->x;
                renderables[index].n = node->n;
                renderables[index].u = vertex->u;
            }
        }

        glBindBuffer(GL_ARRAY_BUFFER, vbo);
        glBufferData(GL_ARRAY_BUFFER, renderables.size() * sizeof(Renderable), renderables.data(), GL_DYNAMIC_DRAW);
    } else {
        if (rebind) {
            glBindBuffer(GL_ARRAY_BUFFER, vbo);
            glBufferData(GL_ARRAY_BUFFER, 3 * facesGpu.size() * sizeof(Renderable), nullptr, GL_DYNAMIC_DRAW);

            CUDA_CHECK(hipGraphicsGLRegisterBuffer(&vboCuda, vbo, hipGraphicsRegisterFlagsWriteDiscard));
        }

        CUDA_CHECK(hipGraphicsMapResources(1, &vboCuda));
        Renderable* renderables;
        size_t nRenderanles;
        CUDA_CHECK(hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&renderables), &nRenderanles, vboCuda));
        updateRenderingDataGpu<<<GRID_SIZE, BLOCK_SIZE>>>(facesGpu.size(), pointer(facesGpu), renderables);
        CUDA_CHECK_LAST();

        CUDA_CHECK(hipGraphicsUnmapResources(1, &vboCuda));
    }
}

void Mesh::bind() {
    glGenVertexArrays(1, &vao);
    glBindVertexArray(vao);
    glGenBuffers(1, &vbo);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(Renderable), reinterpret_cast<void*>(offsetof(Renderable, x)));
    glEnableVertexAttribArray(1);
    glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, sizeof(Renderable), reinterpret_cast<void*>(offsetof(Renderable, n)));
    glEnableVertexAttribArray(2);
    glVertexAttribPointer(2, 2, GL_FLOAT, GL_FALSE, sizeof(Renderable), reinterpret_cast<void*>(offsetof(Renderable, u)));

    updateRenderingData(true);
}

void Mesh::render() const {
    glBindVertexArray(vao);
    glDrawArrays(GL_TRIANGLES, 0, 3 * (!gpu ? faces.size() : facesGpu.size()));
    glBindVertexArray(0);
}

void Mesh::readDataFromFile(const std::string& path) {
    std::ifstream fin(path);
    for (Node* node : nodes)
        fin >> node->x0(0) >> node->x0(1) >> node->x0(2) >> node->x(0) >> node->x(1) >> node->x(2) >> node->v(0) >> node->v(1) >> node->v(2);
    fin.close();
}

void Mesh::writeDataToFile(const std::string& path) {
    std::ofstream fout(path);
    if (!gpu) {
        for (const Node* node : nodes) {
            fout << "v " << node->x(0) << " " << node->x(1) << " " << node->x(2) << std::endl;
            fout << "nv " << node->v(0) << " " << node->v(1) << " " << node->v(2) << std::endl;
        }
        for (const Vertex* vertex : vertices)
            fout << "vt " << vertex->u(0) << " " << vertex->u(1) << std::endl;
        for (const Face* face : faces) {
            fout << "f";
            for (int i = 0; i < 3; i++) {
                Vertex* vertex = face->vertices[i];
                int xIndex = vertex->node->index + 1;
                int uIndex = vertex->index + 1;
                fout << " " << xIndex << "/" << uIndex;
            }
            fout << std::endl;
        }
    } else {
        int nNodes = nodesGpu.size();
        thrust::device_vector<Vector3f> x(nNodes);
        copyX<<<GRID_SIZE, BLOCK_SIZE>>>(nNodes, pointer(nodesGpu), pointer(x));
        CUDA_CHECK_LAST();
        for (const Vector3f& xt : x)
            fout << "v " << xt(0) << " " << xt(1) << " " << xt(2) << std::endl;

        thrust::device_vector<Vector3f> v(nNodes);
        copyX<<<GRID_SIZE, BLOCK_SIZE>>>(nNodes, pointer(nodesGpu), pointer(v));
        CUDA_CHECK_LAST();
        for (const Vector3f& vt : v)
            fout << "nv " << vt(0) << " " << vt(1) << " " << vt(2) << std::endl;

        int nVertices = verticesGpu.size();
        thrust::device_vector<Vector2f> u(nVertices);
        copyU<<<GRID_SIZE, BLOCK_SIZE>>>(nVertices, pointer(verticesGpu), pointer(u));
        CUDA_CHECK_LAST();
        for (const Vector2f& ut : u)
            fout << "vt " << ut(0) << " " << ut(1) << std::endl;
        
        int nFaces = facesGpu.size();
        thrust::device_vector<Pairii> indices(3 * nFaces);
        copyFaceIndices<<<GRID_SIZE, BLOCK_SIZE>>>(nFaces, pointer(facesGpu), pointer(indices));
        CUDA_CHECK_LAST();
        for (int i = 0; i < nFaces; i++) {
            fout << "f";
            for (int j = 0; j < 3; j++) {
                const Pairii& index = indices[3 * i + j];
                fout << " " << index.first + 1 << "/" << index.second + 1;
            }
            fout << std::endl;
        }

        fout.close();
    }
}

void Mesh::printDebugInfo(int selectedFace) {
    if (!gpu) {
        Face* face = faces[selectedFace];
        std::cout << "Nodes=[" << face->vertices[0]->node->index << ", " << face->vertices[1]->node->index << ", " << face->vertices[2]->node->index << "]" << std::endl;
    } else
        printDebugInfoGpu<<<1, 1>>>(pointer(facesGpu), selectedFace);
}
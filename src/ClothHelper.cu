#include "hip/hip_runtime.h"
#include "ClothHelper.cuh"

__global__ void initializeHandles(int nHandles, const int* handleIndices, Node** nodes, Handle* handles) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nHandles; i += nThreads) {
        Node* node = nodes[handleIndices[i]];
        Handle& handle = handles[i];
        node->preserve = true;
        handle.node = node;
        handle.position = node->x;
    }
}

__global__ void collectHandleIndices(int nHandles, const Handle* handles, int* handleIndices) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nHandles; i += nThreads)
        handleIndices[i] = handles[i].node->index;
}

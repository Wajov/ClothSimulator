#include "CudaHelper.cuh"

void cudaCheck(hipError_t err, const char* func, const char* file, int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    }
}

void cudaCheckLast(const char* file, int line) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
    }
}

void cublasCheck(hipblasStatus_t err, const char* func, const char* file, int line) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "CuSPARSE Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << cublasGetStatusString(err) << " " << func << std::endl;
    }
}

void cusparseCheck(hipsparseStatus_t err, const char* func, const char* file, int line) {
    if (err != HIPSPARSE_STATUS_SUCCESS) {
        std::cerr << "CuSPARSE Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipsparseGetErrorString(err) << " " << func << std::endl;
    }
}
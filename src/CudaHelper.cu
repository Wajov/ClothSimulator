#include "CudaHelper.cuh"

void cudaCheck(hipError_t err, const char* func, const char* file, int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    }
}

void cudaCheckLast(const char* file, int line) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
    }
}

void cusparseCheck(hipsparseStatus_t err, const char* func, const char* file, int line) {
    if (err != HIPSPARSE_STATUS_SUCCESS) {
        std::cerr << "CuSPARSE Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipsparseGetErrorString(err) << " " << func << std::endl;
    }
}

void cusolverCheck(hipsolverStatus_t err, const char* func, const char* file, int line) {
    if (err != HIPSOLVER_STATUS_SUCCESS)
        std::cerr << "CuSOLVER Runtime Error at: " << file << ":" << line << std::endl;
}
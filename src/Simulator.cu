#include "Simulator.cuh"

Simulator::Simulator(SimulationMode mode, const std::string& path, const std::string& directory) :
    mode(mode),
    nSteps(0),
    nFrames(0),
    directory(directory) {
    hipDeviceSetLimit(hipLimitMallocHeapSize, 1 << 30);
    if (mode == Simulate || mode == Resume || mode == Replay)
        renderer = new Renderer(900, 900);

    std::ifstream fin(mode == Simulate || mode == SimulateOffline ? path : directory + "/config.json");
    if (!fin.is_open()) {
        std::cerr << "Failed to open configuration file: " << path << std::endl;
        exit(1);
    }
    fin >> json;
    fin.close();

    frameTime = parseFloat(json["frame_time"]);
    frameSteps = parseInt(json["frame_steps"]);
    endTime = parseFloat(json["end_time"], INFINITY);
    dt =  frameTime / frameSteps;

    gravity = parseVector3f(json["gravity"]);
    Wind* windTemp = new Wind();
    if (!gpu)
        wind = windTemp;
    else {
        CUDA_CHECK(hipMalloc(&wind, sizeof(Wind)));
        CUDA_CHECK(hipMemcpy(wind, windTemp, sizeof(Wind), hipMemcpyHostToDevice));
        delete windTemp;
    }
    
    magic = new Magic(json["magic"]);

    motions.resize(json["motions"].size());
    for (int i = 0; i < json["motions"].size(); i++)
        motions[i] = new Motion(json["motions"][i]);
    
    cloths.resize(json["cloths"].size());
    for (int i = 0; i < json["cloths"].size(); i++)
        cloths[i] = new Cloth(json["cloths"][i]);

    obstacles.resize(json["obstacles"].size());
    for (int i = 0; i < json["obstacles"].size(); i++)
        obstacles[i] = new Obstacle(json["obstacles"][i], motions);
}

Simulator::~Simulator() {
    delete magic;
    for (const Motion* motion : motions)
        delete motion;
    for (const Cloth* cloth : cloths)
        delete cloth;
    for (const Obstacle* obstacle : obstacles)
        delete obstacle;
    delete renderer;

    if (!gpu)
        delete wind;
    else
        CUDA_CHECK(hipFree(wind));
}

std::vector<BVH*> Simulator::buildClothBvhs(bool ccd) const {
    std::vector<BVH*> ans(cloths.size());
    for (int i = 0; i < cloths.size(); i++)
        ans[i] = new BVH(cloths[i]->getMesh(), ccd);
    return ans;
}

std::vector<BVH*> Simulator::buildObstacleBvhs(bool ccd) const {
    std::vector<BVH*> ans(obstacles.size());
    for (int i = 0; i < obstacles.size(); i++)
        ans[i] = new BVH(obstacles[i]->getMesh(), ccd);
    return ans;
}

void Simulator::updateBvhs(std::vector<BVH*>& bvhs) const {
    for (BVH* bvh : bvhs)
        bvh->update();
}

void Simulator::destroyBvhs(const std::vector<BVH*>& bvhs) const {
    for (const BVH* bvh : bvhs)
        delete bvh;
}

void Simulator::traverse(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, float thickness, std::function<void(const Face*, const Face*, float)> callback) const {
    for (int i = 0; i < clothBvhs.size(); i++) {
        clothBvhs[i]->traverse(thickness, callback);
        for (int j = 0; j < i; j++)
            clothBvhs[i]->traverse(clothBvhs[j], thickness, callback);
        
        for (int j = 0; j < obstacleBvhs.size(); j++)
            clothBvhs[i]->traverse(obstacleBvhs[j], thickness, callback);
    }
}

thrust::device_vector<Proximity> Simulator::traverse(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, float thickness) const {
    thrust::device_vector<Proximity> ans, proximities;
    for (int i = 0; i < cloths.size(); i++) {
        proximities = std::move(clothBvhs[i]->traverse(thickness));
        ans.insert(ans.end(), proximities.begin(), proximities.end());

        for (int j = 0; j < i; j++) {
            proximities = std::move(clothBvhs[i]->traverse(clothBvhs[j], thickness));
            ans.insert(ans.end(), proximities.begin(), proximities.end());
        }

        for (int j = 0; j < obstacleBvhs.size(); j++) {
            proximities = std::move(clothBvhs[i]->traverse(obstacleBvhs[j], thickness));
            ans.insert(ans.end(), proximities.begin(), proximities.end());
        }
    }
    return ans;
}

void Simulator::updateActive(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, const std::vector<Impact>& impacts) const {
    for (BVH* clothBvh : clothBvhs)
        clothBvh->setAllActive(false);
    for (BVH* obstacleBvh : obstacleBvhs)
        obstacleBvh->setAllActive(false);
    
    for (const Impact& impact : impacts)
        for (int i = 0; i < 4; i++) {
            Node* node = impact.nodes[i];
            for (BVH* clothBvh : clothBvhs)
                if (clothBvh->contain(node))
                    clothBvh->setActive(node, true);
            for (BVH* obstacleBvh : obstacleBvhs)
                if (obstacleBvh->contain(node))
                    obstacleBvh->setActive(node, true);
        }
}

void Simulator::checkImpacts(const Face* face0, const Face* face1, float thickness, std::vector<Impact>& impacts) const {
    Impact impact;
    for (int i = 0; i < 3; i++)
        if (checkVertexFaceImpact(face0->vertices[i], face1, thickness, impact))
            impacts.push_back(impact);
    for (int i = 0; i < 3; i++)
        if (checkVertexFaceImpact(face1->vertices[i], face0, thickness, impact))
            impacts.push_back(impact);
    for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
            if (checkEdgeEdgeImpact(face0->edges[i], face1->edges[j], thickness, impact))
                impacts.push_back(impact);
}

thrust::device_vector<Impact> Simulator::findImpacts(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs) const {
    thrust::device_vector<Proximity> proximities = std::move(traverse(clothBvhs, obstacleBvhs, magic->collisionThickness));
    int nProximities = proximities.size();
    thrust::device_vector<Impact> ans(15 * nProximities);
    checkImpactsGpu<<<GRID_SIZE, BLOCK_SIZE>>>(nProximities, pointer(proximities), magic->collisionThickness, pointer(ans));
    CUDA_CHECK_LAST();

    ans.erase(thrust::remove_if(ans.begin(), ans.end(), IsNull()), ans.end());
    return ans;
}

std::vector<Impact> Simulator::independentImpacts(const std::vector<Impact>& impacts, int deform) const {
    std::vector<Impact> sorted = impacts;
    std::sort(sorted.begin(), sorted.end());
    
    std::unordered_set<Node*> nodes;
    std::vector<Impact> ans;
    for (const Impact& impact : sorted) {
        bool flag = true;
        for (int i = 0; i < 4; i++) {
            Node* node = impact.nodes[i];
            if ((deform == 1 || node->isFree) && nodes.find(node) != nodes.end()) {
                flag = false;
                break;
            }
        }

        if (flag) {
            ans.push_back(impact);
            for (int i = 0; i < 4; i++)
                nodes.insert(impact.nodes[i]);
        }
    }
    return ans;
}

thrust::device_vector<Impact> Simulator::independentImpacts(const thrust::device_vector<Impact>& impacts, int deform) const {
    int nImpacts = impacts.size();
    const Impact* impactsPointer = pointer(impacts);
    int nNodes = 4 * nImpacts;
    thrust::device_vector<Node*> nodes(nNodes), outputNodes(nNodes);
    Node** nodesPointer = pointer(nodes);
    Node** outputNodesPointer = pointer(outputNodes);
    thrust::device_vector<Pairfi> relativeImpacts(nNodes), outputRelativeImpacts(nNodes);
    Pairfi* relativeImpactsPointer = pointer(relativeImpacts);
    Pairfi* outputRelativeImpactsPointer = pointer(outputRelativeImpacts);
    thrust::device_vector<Impact> ans(nImpacts);
    Impact* ansPointer = pointer(ans);
    initializeImpactNodes<<<GRID_SIZE, BLOCK_SIZE>>>(nImpacts, impactsPointer, deform);
    CUDA_CHECK_LAST();

    int num, newNum = nImpacts;
    do {
        num = newNum;
        collectRelativeImpacts<<<GRID_SIZE, BLOCK_SIZE>>>(nImpacts, impactsPointer, deform, nodesPointer, relativeImpactsPointer);
        CUDA_CHECK_LAST();

        thrust::sort_by_key(nodes.begin(), nodes.end(), relativeImpacts.begin());
        auto iter = thrust::reduce_by_key(nodes.begin(), nodes.end(), relativeImpacts.begin(), outputNodes.begin(), outputRelativeImpacts.begin(), thrust::equal_to<Node*>(), thrust::minimum<Pairfi>());

        setImpactMinIndices<<<GRID_SIZE, BLOCK_SIZE>>>(iter.first - outputNodes.begin(), outputRelativeImpactsPointer, outputNodesPointer);
        CUDA_CHECK_LAST();

        checkIndependentImpacts<<<GRID_SIZE, BLOCK_SIZE>>>(nImpacts, impactsPointer, deform, ansPointer);
        CUDA_CHECK_LAST();

        newNum = thrust::count_if(ans.begin(), ans.end(), IsNull());
    } while (num > newNum);

    ans.erase(thrust::remove_if(ans.begin(), ans.end(), IsNull()), ans.end());
    CUDA_CHECK_LAST();

    return ans;
}

void Simulator::updateActive(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, const std::vector<Intersection>& intersections) const {
    for (BVH* clothBvh : clothBvhs)
        clothBvh->setAllActive(false);
    for (BVH* obstacleBvh : obstacleBvhs)
        obstacleBvh->setAllActive(false);
    
    for (const Intersection& intersection : intersections)
        for (int i = 0; i < 3; i++) {
            Node* node0 = intersection.face0->vertices[i]->node;
            for (BVH* clothBvh : clothBvhs)
                if (clothBvh->contain(node0))
                    clothBvh->setActive(node0, true);
            for (BVH* obstacleBvh : obstacleBvhs)
                if (obstacleBvh->contain(node0))
                    obstacleBvh->setActive(node0, true);
            
            Node* node1 = intersection.face1->vertices[i]->node;
            for (BVH* clothBvh : clothBvhs)
                if (clothBvh->contain(node1))
                    clothBvh->setActive(node1, true);
            for (BVH* obstacleBvh : obstacleBvhs)
                if (obstacleBvh->contain(node1))
                    obstacleBvh->setActive(node1, true);
        }
}

Vector3f Simulator::oldPosition(const Vector2f& u, const std::vector<BackupFace>& faces) const {
    for (const BackupFace& face : faces) {
        Vector3f b = face.barycentricCoordinates(u);
        if (b(0) >= -1e-6f && b(1) >= -1e-6f && b(2) >= -1e-5f)
            return face.position(b);
    }
}

Vector3f Simulator::oldPosition(const Face* face, const Vector3f& b, const std::vector<std::vector<BackupFace>>& faces) const {
    if (!face->isFree())
        return face->position(b);
    
    Vector2f u = b(0) * face->vertices[0]->u + b(1) * face->vertices[1]->u + b(2) * face->vertices[2]->u;
    for (int i = 0; i < cloths.size(); i++)
        if (cloths[i]->getMesh()->contain(face))
            return oldPosition(u, faces[i]);
}

void Simulator::checkIntersection(const Face* face0, const Face* face1, std::vector<Intersection>& intersections, const std::vector<std::vector<BackupFace>>& faces) const {
    Intersection intersection;
    Vector3f& b0 = intersection.b0;
    Vector3f& b1 = intersection.b1;
    if (checkIntersectionMidpoint(face0, face1, b0, b1)) {
        intersection.face0 = const_cast<Face*>(face0);
        intersection.face1 = const_cast<Face*>(face1);
        Vector3f x0 = oldPosition(face0, b0, faces);
        Vector3f x1 = oldPosition(face1, b1, faces);
        Vector3f& d = intersection.d;
        d = (x0 - x1).normalized();
        farthestPoint(face0, face1, d, b0, b1);
        intersections.push_back(intersection);
    }
}

thrust::device_vector<Intersection> Simulator::findIntersections(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, const std::vector<thrust::device_vector<BackupFace>>& faces) const {
    thrust::device_vector<Proximity> proximities = std::move(traverse(clothBvhs, obstacleBvhs, magic->collisionThickness));
    int nProximities = proximities.size();
    thrust::device_vector<Intersection> ans(nProximities);
    Intersection* ansPointer = pointer(ans);
    checkIntersectionsGpu<<<GRID_SIZE, BLOCK_SIZE>>>(nProximities, pointer(proximities), ansPointer);
    CUDA_CHECK_LAST();

    ans.erase(thrust::remove_if(ans.begin(), ans.end(), IsNull()), ans.end());
    int nAns = ans.size();
    thrust::device_vector<Vector3f> x(2 * nAns);
    Vector3f* xPointer = pointer(x);
    initializeOldPosition<<<GRID_SIZE, BLOCK_SIZE>>>(nAns, ansPointer, xPointer);
    CUDA_CHECK_LAST();

    for (int i = 0; i < cloths.size(); i++) {
        thrust::device_vector<Vertex*>& vertices = cloths[i]->getMesh()->getVerticesGpu();
        thrust::device_vector<int> indices(2 * nAns);
        int* indicesPointer = pointer(indices);
        thrust::device_vector<Vector2f> u(2 * nAns);
        Vector2f* uPointer = pointer(u);
        collectContainedFaces<<<GRID_SIZE, BLOCK_SIZE>>>(nAns, ansPointer, vertices.size(), pointer(vertices), indicesPointer, uPointer);
        CUDA_CHECK_LAST();

        u.erase(thrust::remove_if(u.begin(), u.end(), indices.begin(), IsNull()), u.end());
        indices.erase(thrust::remove(indices.begin(), indices.end(), -1), indices.end());
        computeOldPosition<<<GRID_SIZE, BLOCK_SIZE>>>(indices.size(), indicesPointer, uPointer, faces[i].size(), pointer(faces[i]), xPointer);
        CUDA_CHECK_LAST();
    }

    computeFarthestPoint<<<GRID_SIZE, BLOCK_SIZE>>>(nAns, xPointer, ansPointer);
    CUDA_CHECK_LAST();

    return ans;
}

void Simulator::obstacleStep() {
    for (Obstacle* obstacle : obstacles)
        obstacle->transform(nSteps * dt);
    
    updateObstacleNodeGeometries();
    updateObstacleFaceGeometries();
}

void Simulator::physicsStep() {
    for (Cloth* cloth : cloths)
        cloth->physicsStep(dt, magic->handleStiffness, gravity, wind);

    updateClothNodeGeometries();
    updateClothFaceGeometries();
}

void Simulator::collisionStep() {
    std::vector<BVH*> clothBvhs = std::move(buildClothBvhs(true));
    std::vector<BVH*> obstacleBvhs = std::move(buildObstacleBvhs(true));
    int deform;
    float obstacleMass = 1e3f;

    if (!gpu) {
        std::vector<Impact> impacts;
        for (deform = 0; deform < 2; deform++) {
            impacts.clear();
            bool success = false;
            for (int i = 0; i < MAX_COLLISION_ITERATION; i++) {
                if (!impacts.empty())
                    updateActive(clothBvhs, obstacleBvhs, impacts);
                
                std::vector<Impact> newImpacts;
                traverse(clothBvhs, obstacleBvhs, magic->collisionThickness, [&](const Face* face0, const Face* face1, float thickness) {
                    checkImpacts(face0, face1, thickness, newImpacts);
                });
                if (newImpacts.empty()) {
                    success = true;
                    break;
                }

                newImpacts = std::move(independentImpacts(newImpacts, deform));
                impacts.insert(impacts.end(), newImpacts.begin(), newImpacts.end());
                Optimization* optimization = new CollisionOptimization(impacts, magic->collisionThickness, deform, obstacleMass);
                optimization->solve();
                delete optimization;

                updateBvhs(clothBvhs);
                if (deform == 1) {
                    updateBvhs(obstacleBvhs);
                    obstacleMass *= 0.5f;
                }
            }
            if (success)
                break;
        }
    } else {
        thrust::device_vector<Impact> impacts;
        for (deform = 0; deform < 2; deform++) {
            impacts.clear();
            bool success = false;
            for (int i = 0; i < MAX_COLLISION_ITERATION; i++) {
                thrust::device_vector<Impact> newImpacts = std::move(findImpacts(clothBvhs, obstacleBvhs));
                if (newImpacts.empty()) {
                    success = true;
                    break;
                }

                newImpacts = std::move(independentImpacts(newImpacts, deform));
                impacts.insert(impacts.end(), newImpacts.begin(), newImpacts.end());
                Optimization* optimization = new CollisionOptimization(impacts, magic->collisionThickness, deform, obstacleMass);
                optimization->solve();
                delete optimization;

                updateBvhs(clothBvhs);
                if (deform == 1) {
                    updateBvhs(obstacleBvhs);
                    obstacleMass *= 0.5f;
                }
            }
            if (success)
                break;
        }
    }

    destroyBvhs(clothBvhs);
    destroyBvhs(obstacleBvhs);

    updateClothNodeGeometries();
    updateClothFaceGeometries();
    updateVelocities();
    if (deform == 1) {
        updateObstacleNodeGeometries();
        updateObstacleFaceGeometries();
    }
}

void Simulator::remeshingStep() {
    std::vector<BVH*> obstacleBvhs = std::move(buildObstacleBvhs(false));
    for (Cloth* cloth : cloths)
        cloth->remeshingStep(obstacleBvhs, 10.0f * magic->repulsionThickness);

    destroyBvhs(obstacleBvhs);

    updateStructures();
    updateClothNodeGeometries();
    updateClothFaceGeometries();
}

void Simulator::separationStep(const std::vector<std::vector<BackupFace>>& faces) {
    std::vector<BVH*> clothBvhs = std::move(buildClothBvhs(false));
    std::vector<BVH*> obstacleBvhs = std::move(buildObstacleBvhs(false));
    int deform;
    float obstacleArea = 1e3f;

    std::vector<Intersection> intersections;
    for (deform = 0; deform < 2; deform++) {
        intersections.clear();
        bool success = false;
        for (int i = 0; i < MAX_SEPARATION_ITERATION; i++) {
            if (!intersections.empty())
                updateActive(clothBvhs, obstacleBvhs, intersections);
            
            std::vector<Intersection> newIntersections;
            traverse(clothBvhs, obstacleBvhs, magic->collisionThickness, [&](const Face* face0, const Face* face1, float thickness) {
                checkIntersection(face0, face1, newIntersections, faces);
            });
            if (newIntersections.empty()) {
                success = true;
                break;
            }

            intersections.insert(intersections.end(), newIntersections.begin(), newIntersections.end());
            Optimization* optimization = new SeparationOptimization(intersections, magic->collisionThickness, deform, obstacleArea);
            optimization->solve();
            delete optimization;

            updateClothFaceGeometries();
            updateBvhs(clothBvhs);
            if (deform == 1) {
                updateObstacleFaceGeometries();
                updateBvhs(obstacleBvhs);
                obstacleArea *= 0.5f;
            }
        }
        if (success)
            break;
    }

    destroyBvhs(clothBvhs);
    destroyBvhs(obstacleBvhs);

    updateClothNodeGeometries();
    updateVelocities();
    if (deform == 1)
        updateObstacleNodeGeometries();
}

void Simulator::separationStep(const std::vector<thrust::device_vector<BackupFace>>& faces) {
    std::vector<BVH*> clothBvhs = std::move(buildClothBvhs(false));
    std::vector<BVH*> obstacleBvhs = std::move(buildObstacleBvhs(false));
    int deform;
    float obstacleArea = 1e3f;

    thrust::device_vector<Intersection> intersections;
    for (deform = 0; deform < 2; deform++) {
        intersections.clear();
        bool success = false;
        for (int i = 0; i < MAX_SEPARATION_ITERATION; i++) {
            thrust::device_vector<Intersection> newIntersections = std::move(findIntersections(clothBvhs, obstacleBvhs, faces));
            if (newIntersections.empty()) {
                success = true;
                break;
            }

            intersections.insert(intersections.end(), newIntersections.begin(), newIntersections.end());
            Optimization* optimization = new SeparationOptimization(intersections, magic->collisionThickness, deform, obstacleArea);
            optimization->solve();
            delete optimization;

            updateClothFaceGeometries();
            updateBvhs(clothBvhs);
            if (deform == 1) {
                updateObstacleFaceGeometries();
                updateBvhs(obstacleBvhs);
                obstacleArea *= 0.5f;
            }
        }
        if (success)
            break;
    }

    destroyBvhs(clothBvhs);
    destroyBvhs(obstacleBvhs);

    updateClothNodeGeometries();
    updateVelocities();
    if (deform == 1)
        updateObstacleNodeGeometries();
}

void Simulator::updateStructures() {
    for (Cloth* cloth : cloths)
        cloth->getMesh()->updateStructures();
}

void Simulator::updateClothNodeGeometries() {
    for (Cloth* cloth : cloths)
        cloth->getMesh()->updateNodeGeometries();
}

void Simulator::updateObstacleNodeGeometries() {
    for (Obstacle* obstacle : obstacles)
        obstacle->getMesh()->updateNodeGeometries();
}

void Simulator::updateClothFaceGeometries() {
    for (Cloth* cloth : cloths)
        cloth->getMesh()->updateFaceGeometries();
}

void Simulator::updateObstacleFaceGeometries() {
    for (Obstacle* obstacle : obstacles)
        obstacle->getMesh()->updateFaceGeometries();
}

void Simulator::updateVelocities() {
    for (Cloth* cloth : cloths)
        cloth->getMesh()->updateVelocities(dt);
}

void Simulator::updateRenderingData(bool rebind) {
    for (Cloth* cloth : cloths)
        cloth->getMesh()->updateRenderingData(rebind);
    for (Obstacle* obstacle : obstacles)
        obstacle->getMesh()->updateRenderingData(false);
}

void Simulator::simulateStep(bool offline) {
    if ((++nSteps) % frameSteps == 0)
        nFrames++;
    std::cout << "Frame [" << nFrames << "], Step [" << nSteps << "]:" << std::endl;

    obstacleStep();

    std::chrono::duration<float> d;
    auto t0 = std::chrono::high_resolution_clock::now();
    
    physicsStep();
    auto t1 = std::chrono::high_resolution_clock::now();
    d = t1 - t0;
    std::cout << "Physics Step: " << d.count() << "s";
    
    collisionStep();
    auto t2 = std::chrono::high_resolution_clock::now();
    d = t2 - t1;
    std::cout << ", Collision Step: " << d.count() << "s";
    
    if (nSteps % frameSteps == 0) {
        if (!gpu) {
            std::vector<std::vector<BackupFace>> faces(cloths.size());
            for (int i = 0; i < cloths.size(); i++)
                faces[i] = std::move(cloths[i]->getMesh()->backupFaces());

            remeshingStep();
            auto t3 = std::chrono::high_resolution_clock::now();
            d = t3 - t2;
            std::cout << ", Remeshing Step: " << d.count() << "s";

            separationStep(faces);
            auto t4 = std::chrono::high_resolution_clock::now();
            d = t4 - t3;
            std::cout << ", Separation Step: " << d.count() << "s";
        } else {
            std::vector<thrust::device_vector<BackupFace>> faces(cloths.size());
            for (int i = 0; i < cloths.size(); i++)
                faces[i] = std::move(cloths[i]->getMesh()->backupFacesGpu());

            remeshingStep();
            auto t3 = std::chrono::high_resolution_clock::now();
            d = t3 - t2;
            std::cout << ", Remeshing Step: " << d.count() << "s";

            separationStep(faces);
            auto t4 = std::chrono::high_resolution_clock::now();
            d = t4 - t3;
            std::cout << ", Separation Step: " << d.count() << "s";
        }

        if (!offline)
            updateRenderingData(true);
        else
            save();
    } else if (!offline)
        updateRenderingData(false);

    std::cout << std::endl;
}

void Simulator::replayStep() {
    if (nFrames == endFrame)
        return;

    nSteps += frameSteps;
    nFrames++;
    std::cout << "Frame [" << nFrames << "]" << std::endl;

    if (load()) {
        obstacleStep();
        updateRenderingData(true);
    }
}

void Simulator::bind() {
    for (Cloth* cloth : cloths)
        cloth->bind();
    for (Obstacle* obstacle : obstacles)
        obstacle->bind();
}

void Simulator::render() const {
    Vector3f lightDirection = renderer->getLightDirection();
    Vector3f cameraPosition = renderer->getCameraPosition();
    Matrix4x4f model = renderer->getModel();
    Matrix4x4f view = renderer->getView();
    Matrix4x4f projection = renderer->getProjection();

    for (const Cloth* cloth : cloths)
        cloth->render(model, view, projection, cameraPosition, lightDirection);
    
    for (const Obstacle* obstacle : obstacles)
        obstacle->render(model, view, projection, cameraPosition, lightDirection);
}

bool Simulator::load() {
    for (int i = 0; i < cloths.size(); i++) {
        std::string path = directory + "/frame" + std::to_string(nFrames) + "_cloth" + std::to_string(i) + ".obj";
        if (!std::filesystem::exists(path))
            return false;
    }

    for (int i = 0; i < cloths.size(); i++) {
        std::string path = directory + "/frame" + std::to_string(nFrames) + "_cloth" + std::to_string(i) + ".obj";
        cloths[i]->load(path);
    }
}

void Simulator::save() {
    for (int i = 0; i < cloths.size(); i++)
        cloths[i]->save(directory + "/frame" + std::to_string(nFrames) + "_cloth" + std::to_string(i) + ".obj", json["cloths"][i]);

    std::ofstream fout(directory + "/config.json");
    fout << json;
}

int Simulator::lastFrame() const {
    int ans = 0;
    bool flag;
    do {
        ans++;
        flag = true;
        for (int i = 0; i < cloths.size(); i++)
            if (!std::filesystem::exists(directory + "/frame" + std::to_string(ans) + "_cloth" + std::to_string(i) + ".obj")) {
                flag = false;
                break;
            }
    } while (flag);

    return ans - 1;
}

bool Simulator::finished() const {
    return (nSteps + 1) * dt >= endTime;
}

void Simulator::simulate() {
    if (!gpu) {
        std::vector<std::vector<BackupFace>> faces(cloths.size());
        for (int i = 0; i < cloths.size(); i++)
            faces[i] = std::move(cloths[i]->getMesh()->backupFaces());

        remeshingStep();
        separationStep(faces);
    } else {
        std::vector<thrust::device_vector<BackupFace>> faces(cloths.size());
        for (int i = 0; i < cloths.size(); i++)
            faces[i] = std::move(cloths[i]->getMesh()->backupFacesGpu());

        remeshingStep();
        separationStep(faces);
    }
    bind();

    while (!glfwWindowShouldClose(renderer->getWindow()) && !finished()) {
        glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        render();
        if (!renderer->getPause())
            simulateStep(false);                

        glfwSwapBuffers(renderer->getWindow());
        glfwPollEvents();
    }
}

void Simulator::simulateOffline() {
    if (!std::filesystem::is_directory(directory))
        std::filesystem::create_directories(directory);

    if (!gpu) {
        std::vector<std::vector<BackupFace>> faces(cloths.size());
        for (int i = 0; i < cloths.size(); i++)
            faces[i] = std::move(cloths[i]->getMesh()->backupFaces());

        remeshingStep();
        separationStep(faces);
    } else {
        std::vector<thrust::device_vector<BackupFace>> faces(cloths.size());
        for (int i = 0; i < cloths.size(); i++)
            faces[i] = std::move(cloths[i]->getMesh()->backupFacesGpu());

        remeshingStep();
        separationStep(faces);
    }
    save();

    while (!finished())
        simulateStep(true);
}

void Simulator::resume() {
    nFrames = lastFrame();
    nSteps = nFrames * frameSteps;
    obstacleStep();
    bind();
    
    while (!glfwWindowShouldClose(renderer->getWindow()) && !finished()) {
        glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        render();
        if (!renderer->getPause())
            simulateStep(false);

        glfwSwapBuffers(renderer->getWindow());
        glfwPollEvents();
    }
}

void Simulator::resumeOffline() {
    nFrames = lastFrame();
    nSteps = nFrames * frameSteps;
    obstacleStep();

    while (!finished())
        simulateStep(true);
}

void Simulator::replay() {
    endFrame = lastFrame();
    load();
    bind();

    while (!glfwWindowShouldClose(renderer->getWindow())) {
        auto t0 = std::chrono::high_resolution_clock::now();
       
        glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        render();
        if (!renderer->getPause())
            replayStep();

        glfwSwapBuffers(renderer->getWindow());
        glfwPollEvents();

        std::chrono::duration<float> d;
        do {
            auto t1 = std::chrono::high_resolution_clock::now();
            d = t1 - t0;
        } while (d.count() < frameTime);
    }
}

void Simulator::start() {
    switch (mode) {
    case Simulate:
        simulate();
        break;
    case SimulateOffline:
        simulateOffline();
        break;
    case Resume:
        resume();
        break;
    case ResumeOffline:
        resumeOffline();
        break;
    case Replay:
        replay();
        break;
    }
}
#include "Simulator.cuh"

Simulator::Simulator(const std::string& path) :
    MAX_ITERATION(30),
    nSteps(0),
    selectedCloth(-1) {
    std::ifstream fin(path);
    if (!fin.is_open()) {
        std::cerr << "Failed to open configuration file: " << path << std::endl;
        exit(1);
    }

    Json::Value json;
    fin >> json;

    frameTime = parseFloat(json["frame_time"]);
    frameSteps = parseInt(json["frame_steps"]);
    dt =  frameTime / frameSteps;

    gravity = parseVector3f(json["gravity"]);
    wind = new Wind();

    magic = new Magic(json["magic"]);
    
    for (const Json::Value& clothJson : json["cloths"])
        cloths.push_back(new Cloth(clothJson));

    for (const Json::Value& obstacleJson : json["obstacles"])
        obstacles.push_back(new Obstacle(obstacleJson));

    fin.close();

    // cloths[0]->readDataFromFile("input.txt");
    remeshingStep();
    bind();

    glGenFramebuffers(1, &fbo);
    glBindFramebuffer(GL_FRAMEBUFFER, fbo);
    glGenTextures(1, &indexTexture);
    glGenRenderbuffers(1, &rbo);
    glBindFramebuffer(GL_FRAMEBUFFER, 0);

    indexShader = new Shader("shader/Vertex.glsl", "shader/IndexFragment.glsl");

    if (gpu) {
        CUDA_CHECK(hipMalloc(&windGpu, sizeof(Wind)));
        CUDA_CHECK(hipMemcpy(windGpu, wind, sizeof(Wind), hipMemcpyHostToDevice));
    }
}

Simulator::~Simulator() {
    delete magic;
    delete wind;
    for (const Cloth* cloth : cloths)
        delete cloth;
    for (const Obstacle* obstacle : obstacles)
        delete obstacle;
    delete indexShader;

    if (gpu)
        CUDA_CHECK(hipFree(windGpu));
}

void Simulator::updateActive(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, const std::vector<ImpactZone*>& zones) const {
    for (BVH* clothBvh : clothBvhs)
        clothBvh->setAllActive(false);
    for (BVH* obstacleBvh : obstacleBvhs)
        obstacleBvh->setAllActive(false);
    
    for (ImpactZone* zone : zones) {
        if (!zone->getActive())
            continue;
        std::vector<Vertex*>& vertices = zone->getVertices();
        for (const Vertex* vertex : vertices) {
            for (BVH* clothBvh : clothBvhs)
                if (clothBvh->contain(vertex))
                    clothBvh->setActive(vertex, true);
            for (BVH* obstacleBvh : obstacleBvhs)
                if (obstacleBvh->contain(vertex))
                    obstacleBvh->setActive(vertex, true);
        }
    }
}

void Simulator::traverse(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, float thickness, std::function<void(const Face*, const Face*, float)> callback) {
    for (int i = 0; i < clothBvhs.size(); i++) {
        clothBvhs[i]->traverse(thickness, callback);
        for (int j = 0; j < i; j++)
            clothBvhs[i]->traverse(clothBvhs[j], thickness, callback);
        
        for (int j = 0; j < obstacleBvhs.size(); j++)
            clothBvhs[i]->traverse(obstacleBvhs[j], thickness, callback);
    }
}

std::vector<Impact> Simulator::independentImpacts(const std::vector<Impact>& impacts) const {
    std::vector<Impact> sorted = impacts;
    std::sort(sorted.begin(), sorted.end());
    
    std::unordered_set<Vertex*> vertices;
    std::vector<Impact> ans;
    for (const Impact& impact : sorted) {
        bool flag = true;
        for (int i = 0; i < 4; i++)
            if (impact.vertices[i]->isFree && vertices.find(impact.vertices[i]) != vertices.end()) {
                flag = false;
                break;
            }
        if (flag) {
            ans.push_back(impact);
            for (int i = 0; i < 4; i++)
                vertices.insert(impact.vertices[i]);
        }
    }
    return ans;
}

ImpactZone* Simulator::findImpactZone(const Vertex* vertex, std::vector<ImpactZone*>& zones) const {
    for (ImpactZone* zone : zones)
        if (zone->contain(vertex))
            return zone;

    ImpactZone* zone = new ImpactZone();
    zone->addVertex(vertex);
    zones.push_back(zone);
    return zone;
}

void Simulator::addImpacts(const std::vector<Impact>& impacts, std::vector<ImpactZone*>& zones, bool deformObstacles) const {
    for (ImpactZone* zone : zones)
        zone->setActive(false);
    
    for (const Impact& impact : impacts) {
        ImpactZone* zone = nullptr;
        for (int i = 0; i < 4; i++) {
            Vertex* vertex = impact.vertices[i];
            if (vertex->isFree || deformObstacles) {
                if (zone == nullptr)
                    zone = findImpactZone(vertex, zones);
                else {
                    ImpactZone* zoneTemp = findImpactZone(vertex, zones);
                    if (zone != zoneTemp) {
                        zone->merge(zoneTemp);
                        zones.erase(std::remove(zones.begin(), zones.end(), zoneTemp), zones.end());
                        delete zoneTemp;
                    }
                }
            }
        }
        zone->addImpact(impact);
        zone->setActive(true);
    }
}

void Simulator::resetObstacles() {
    for (Obstacle* obstacle : obstacles)
        obstacle->reset();
}

void Simulator::physicsStep() {
    for (Cloth* cloth : cloths)
        cloth->physicsStep(dt, magic->handleStiffness, gravity, !gpu ? wind : windGpu);
    updateGeometries();
}

void Simulator::collisionStep() {
    if (!gpu) {
        std::vector<BVH*> clothBvhs, obstacleBvhs;
        for (const Cloth* cloth : cloths)
            clothBvhs.push_back(new BVH(cloth->getMesh(), true));
        for (const Obstacle* obstacle: obstacles)
            obstacleBvhs.push_back(new BVH(obstacle->getMesh(), true));
        
        std::vector<ImpactZone*> zones;
        float obstacleMass = 1e3f;
        for (int deform = 0; deform < 2; deform++) {
            zones.clear();
            bool success = false;
            for (int i = 0; i < MAX_ITERATION; i++) {
                if (!zones.empty())
                    updateActive(clothBvhs, obstacleBvhs, zones);
                
                std::vector<Impact> impacts;
                traverse(clothBvhs, obstacleBvhs, magic->collisionThickness, [&](const Face* face0, const Face* face1, float thickness) {
                    checkImpacts(face0, face1, thickness, impacts);
                });
                impacts = std::move(independentImpacts(impacts));
                if (impacts.empty()) {
                    success = true;
                    break;
                }

                addImpacts(impacts, zones, deform == 1);
                for (const ImpactZone* zone : zones)
                    if (zone->getActive()) {
                        Optimization* optimization = new ImpactZoneOptimization(zone, magic->collisionThickness, obstacleMass);
                        augmentedLagrangianMethod(optimization);
                        delete optimization;
                    }

                for (BVH* clothBvh : clothBvhs)
                    clothBvh->update();
                for (BVH* obstacleBvh : obstacleBvhs)
                    obstacleBvh->update();
                if (deform == 1)
                    obstacleMass *= 0.5f;
            }
            if (success)
                break;
        }

        updateGeometries();
        updateVelocities();

        for (const BVH* clothBvh : clothBvhs)
            delete clothBvh;
        for (const BVH* obstacleBvh : obstacleBvhs)
            delete obstacleBvh;
        for (const ImpactZone* zone : zones)
            delete zone;
    } else {
        // TODO
    }
}

void Simulator::remeshingStep() {
    if (!gpu) {
        std::vector<BVH*> obstacleBvhs;
        for (const Obstacle* obstacle: obstacles)
            obstacleBvhs.push_back(new BVH(obstacle->getMesh(), false));

        for (Cloth* cloth : cloths)
            cloth->remeshingStep(obstacleBvhs, 10.0f * magic->repulsionThickness);

        updateIndices();
        updateGeometries();

        for (const BVH* obstacleBvh : obstacleBvhs)
            delete obstacleBvh;
    } else {
        // TODO
    }
}

void Simulator::updateGeometries() {
    for (Cloth* cloth : cloths)
        cloth->updateGeometries();
}

void Simulator::updateVelocities() {
    for (Cloth* cloth : cloths)
        cloth->updateVelocities(dt);
}

void Simulator::updateIndices() {
    for (Cloth* cloth : cloths)
        cloth->updateIndices();
}

void Simulator::updateRenderingData(bool rebind) {
    for (Cloth* cloth : cloths)
        cloth->updateRenderingData(rebind);
}

void Simulator::bind() {
    for (Cloth* cloth : cloths)
        cloth->bind();
    for (Obstacle* obstacle : obstacles)
        obstacle->bind();
}

void Simulator::render(int width, int height, const Matrix4x4f& model, const Matrix4x4f& view, const Matrix4x4f& projection, const Vector3f& cameraPosition, const Vector3f& lightDirection) const {
    glBindFramebuffer(GL_DRAW_FRAMEBUFFER, fbo);
    glBindTexture(GL_TEXTURE_2D, indexTexture);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RG32I, width, height, 0, GL_RG_INTEGER, GL_INT, nullptr);
    glBindTexture(GL_TEXTURE_2D, 0);
    glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, indexTexture, 0);
    glBindRenderbuffer(GL_RENDERBUFFER, rbo);
    glRenderbufferStorage(GL_RENDERBUFFER, GL_DEPTH_COMPONENT32F, width, height);
    glBindRenderbuffer(GL_RENDERBUFFER, 0);
    glFramebufferRenderbuffer(GL_FRAMEBUFFER, GL_DEPTH_ATTACHMENT, GL_RENDERBUFFER, rbo);
    int color[2] = {-1, -1};
    glClearBufferiv(GL_COLOR, 0, color);
    glClear(GL_DEPTH_BUFFER_BIT);
    indexShader->use();
    indexShader->setMat4("model", model);
    indexShader->setMat4("view", view);
    indexShader->setMat4("projection", projection);
    for (int i = 0; i < cloths.size(); i++) {
        indexShader->setInt("clothIndex", i);
        cloths[i]->getMesh()->renderFaces();
    }
    glBindFramebuffer(GL_FRAMEBUFFER, 0);

    for (int i = 0; i < cloths.size(); i++)
        cloths[i]->render(model, view, projection, cameraPosition, lightDirection, selectedCloth == i ? selectedFace : -1);
    
    // for (const Obstacle* obstacle : obstacles)
    //     obstacle->render(model, view, projection, cameraPosition, lightDirection);
}

void Simulator::step() {
    nSteps++;
    std::cout << "Step [" << nSteps << "]:" << std::endl;

    selectedCloth = -1;

    resetObstacles();
    
    std::chrono::duration<float> d;
    auto t0 = std::chrono::high_resolution_clock::now();
    
    physicsStep();
    auto t1 = std::chrono::high_resolution_clock::now();
    d = t1 - t0;
    std::cout << "Physics Step: " << d.count() << "s";
    
    collisionStep();
    auto t2 = std::chrono::high_resolution_clock::now();
    d = t2 - t1;
    std::cout << ", Collision Step: " << d.count() << "s";
    
    if (nSteps % frameSteps == 0) {
        remeshingStep();
        auto t3 = std::chrono::high_resolution_clock::now();
        d = t3 - t2;
        std::cout << ", Remeshing Step: " << d.count() << "s";
        updateRenderingData(true);
    } else
        updateRenderingData(false);

    std::cout << std::endl;
}

void Simulator::printDebugInfo(int x, int y) {
    glBindFramebuffer(GL_READ_FRAMEBUFFER, fbo);
    glReadBuffer(GL_COLOR_ATTACHMENT0);

    Pixel pixel;
    glReadPixels(x, y, 1, 1, GL_RG_INTEGER, GL_INT, &pixel);
    selectedCloth = pixel.clothIndex;
    selectedFace = pixel.faceInedx;
    if (selectedCloth != -1 && selectedFace != -1)
        cloths[selectedCloth]->printDebugInfo(selectedFace);

    glReadBuffer(GL_NONE);
    glBindFramebuffer(GL_FRAMEBUFFER, 0);
}

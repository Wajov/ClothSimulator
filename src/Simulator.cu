#include "hip/hip_runtime.h"
#include "Simulator.cuh"

Simulator::Simulator(SimulationMode mode, const std::string& path, const std::string& directory) :
    mode(mode),
    nSteps(0),
    nFrames(0),
    directory(directory) {
    hipDeviceSetLimit(hipLimitMallocHeapSize, 1 << 30);
    if (mode == Simulate || mode == Resume || mode == Replay)
        renderer = new Renderer(900, 900);

    std::ifstream fin(mode == Simulate || mode == SimulateOffline ? path : directory + "/config.json");
    if (!fin.is_open()) {
        std::cerr << "Failed to open configuration file: " << path << std::endl;
        exit(1);
    }
    fin >> json;
    fin.close();

    frameTime = parseFloat(json["frame_time"]);
    frameSteps = parseInt(json["frame_steps"]);
    endTime = parseFloat(json["end_time"], INFINITY);
    endFrame = parseInt(json["end_frame"], INT_MAX);
    dt =  frameTime / frameSteps;

    clothFriction = parseFloat(json["friction"], 0.6f);
    obstacleFriction = parseFloat(json["obs_friction"], 0.3f);
    gravity = parseVector3f(json["gravity"]);
    Wind* windTemp = new Wind();
    if (!gpu)
        wind = windTemp;
    else {
        CUDA_CHECK(hipMalloc(&wind, sizeof(Wind)));
        CUDA_CHECK(hipMemcpy(wind, windTemp, sizeof(Wind), hipMemcpyHostToDevice));
        delete windTemp;
    }
    
    magic = new Magic(json["magic"]);

    motions.resize(json["motions"].size());
    for (int i = 0; i < json["motions"].size(); i++)
        motions[i] = new Motion(json["motions"][i]);
    
    cloths.resize(json["cloths"].size());
    for (int i = 0; i < json["cloths"].size(); i++)
        cloths[i] = new Cloth(json["cloths"][i]);

    obstacles.resize(json["obstacles"].size());
    for (int i = 0; i < json["obstacles"].size(); i++)
        obstacles[i] = new Obstacle(json["obstacles"][i], motions);
}

Simulator::~Simulator() {
    delete magic;
    for (const Motion* motion : motions)
        delete motion;
    for (const Cloth* cloth : cloths)
        delete cloth;
    for (const Obstacle* obstacle : obstacles)
        delete obstacle;
    delete renderer;

    if (!gpu)
        delete wind;
    else
        CUDA_CHECK(hipFree(wind));
}

std::vector<BVH*> Simulator::buildClothBvhs(bool ccd) const {
    std::vector<BVH*> ans(cloths.size());
    for (int i = 0; i < cloths.size(); i++)
        ans[i] = new BVH(cloths[i]->getMesh(), ccd);
    return ans;
}

std::vector<BVH*> Simulator::buildObstacleBvhs(bool ccd) const {
    std::vector<BVH*> ans(obstacles.size());
    for (int i = 0; i < obstacles.size(); i++)
        ans[i] = new BVH(obstacles[i]->getMesh(), ccd);
    return ans;
}

void Simulator::updateBvhs(std::vector<BVH*>& bvhs) const {
    for (BVH* bvh : bvhs)
        bvh->update();
}

void Simulator::destroyBvhs(const std::vector<BVH*>& bvhs) const {
    for (const BVH* bvh : bvhs)
        delete bvh;
}

void Simulator::traverse(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, float thickness, std::function<void(const Face*, const Face*, float)> callback) const {
    for (int i = 0; i < clothBvhs.size(); i++) {
        clothBvhs[i]->traverse(thickness, callback);
        for (int j = 0; j < i; j++)
            clothBvhs[i]->traverse(clothBvhs[j], thickness, callback);
        
        for (int j = 0; j < obstacleBvhs.size(); j++)
            clothBvhs[i]->traverse(obstacleBvhs[j], thickness, callback);
    }
}

thrust::device_vector<PairFF> Simulator::traverse(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, float thickness) const {
    thrust::device_vector<PairFF> ans, pairs;
    for (int i = 0; i < cloths.size(); i++) {
        pairs = std::move(clothBvhs[i]->traverse(thickness));
        ans.insert(ans.end(), pairs.begin(), pairs.end());

        for (int j = 0; j < i; j++) {
            pairs = std::move(clothBvhs[i]->traverse(clothBvhs[j], thickness));
            ans.insert(ans.end(), pairs.begin(), pairs.end());
        }

        for (int j = 0; j < obstacleBvhs.size(); j++) {
            pairs = std::move(clothBvhs[i]->traverse(obstacleBvhs[j], thickness));
            ans.insert(ans.end(), pairs.begin(), pairs.end());
        }
    }
    return ans;
}

void Simulator::checkVertexFaceProximity(const Vertex* vertex, const Face* face, std::unordered_map<PairNi, PairfF, PairHash>& nodeProximities, std::unordered_map<PairFi, PairfN, PairHash>& faceProximities) const {
    Node* node = vertex->node;
    Node* node0 = face->vertices[0]->node;
    Node* node1 = face->vertices[1]->node;
    Node* node2 = face->vertices[2]->node;
    if (node == node0 || node == node1 || node == node2)
        return;
    
    Vector3f n;
    float w[4];
    float d = abs(signedVertexFaceDistance(node->x, node0->x, node1->x, node2->x, n, w));
    bool inside = (min(-w[1], -w[2], -w[3]) >= 1e-6f);
    if (!inside)
        return;

    if (node->isFree) {
        int side = n.dot(node->n) >= 0.0f ? 0 : 1;
        PairNi key(node, side);
        PairfF value(d, const_cast<Face*>(face));
        if (nodeProximities.find(key) == nodeProximities.end())
            nodeProximities[key] = value;
        else
            nodeProximities[key] = min(nodeProximities[key], value);
    }
    if (face->isFree()) {
        int side = -n.dot(face->n) >= 0.0f ? 0 : 1;
        PairFi key(const_cast<Face*>(face), side);
        PairfN value(d, node);
        if (faceProximities.find(key) == faceProximities.end())
            faceProximities[key] = value;
        else
            faceProximities[key] = min(faceProximities[key], value);
    }
}

void Simulator::checkEdgeEdgeProximity(const Edge* edge0, const Edge* edge1, std::unordered_map<PairEi, PairfE, PairHash>& edgeProximities) const {
    Node* node0 = edge0->nodes[0];
    Node* node1 = edge0->nodes[1];
    Node* node2 = edge1->nodes[0];
    Node* node3 = edge1->nodes[1];
    if (node0 == node2 || node0 == node3 || node1 == node2 || node1 == node3)
        return;
    
    Vector3f n;
    float w[4];
    float d = abs(signedEdgeEdgeDistance(node0->x, node1->x, node2->x, node3->x, n, w));
    bool inside = (min(w[0], w[1], -w[2], -w[3]) >= 1e-6f && inEdge(w[1], edge0, edge1) && inEdge(-w[3], edge1, edge0));
    if (!inside)
        return;
    
    if (edge0->isFree()) {
        int side = n.dot(edge0->nodes[0]->n + edge0->nodes[1]->n) >= 0.0f ? 0 : 1;
        PairEi key(const_cast<Edge*>(edge0), side);
        PairfE value(d, const_cast<Edge*>(edge1));
        if (edgeProximities.find(key) == edgeProximities.end())
            edgeProximities[key] = value;
        else
            edgeProximities[key] = min(edgeProximities[key], value);
    }
    if (edge1->isFree()) {
        int side = -n.dot(edge1->nodes[0]->n + edge1->nodes[1]->n) >= 0.0f ? 0 : 1;
        PairEi key(const_cast<Edge*>(edge1), side);
        PairfE value(d, const_cast<Edge*>(edge0));
        if (edgeProximities.find(key) == edgeProximities.end())
            edgeProximities[key] = value;
        else
            edgeProximities[key] = min(edgeProximities[key], value);
    }
}

void Simulator::checkProximities(const Face* face0, const Face* face1, float thickness, std::unordered_map<PairNi, PairfF, PairHash>& nodeProximities, std::unordered_map<PairEi, PairfE, PairHash>& edgeProximities, std::unordered_map<PairFi, PairfN, PairHash>& faceProximities) const {
    for (int i = 0; i < 3; i++)
        checkVertexFaceProximity(face0->vertices[i], face1, nodeProximities, faceProximities);
    for (int i = 0; i < 3; i++)
        checkVertexFaceProximity(face1->vertices[i], face0, nodeProximities, faceProximities);
    for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
            checkEdgeEdgeProximity(face0->edges[i], face1->edges[j], edgeProximities);
}

std::vector<Proximity> Simulator::findProximities(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs) const {
    std::unordered_map<PairNi, PairfF, PairHash> nodeProximities;
    std::unordered_map<PairEi, PairfE, PairHash> edgeProximities;
    std::unordered_map<PairFi, PairfN, PairHash> faceProximities;
    traverse(clothBvhs, obstacleBvhs, 2.0f * magic->repulsionThickness, [&](const Face* face0, const Face* face1, float thickness) {
        checkProximities(face0, face1, thickness, nodeProximities, edgeProximities, faceProximities);
    });

    std::vector<Proximity> ans;
    for (const std::pair<PairNi, PairfF>& pair : nodeProximities)
        if (pair.second.first < 2.0f * magic->repulsionThickness)
            ans.emplace_back(pair.first.first, pair.second.second, magic->collisionStiffness, clothFriction, obstacleFriction);
    for (const std::pair<PairEi, PairfE>& pair : edgeProximities)
        if (pair.second.first < 2.0f * magic->repulsionThickness)
            ans.emplace_back(pair.first.first, pair.second.second, magic->collisionStiffness, clothFriction, obstacleFriction);
    for (const std::pair<PairFi, PairfN>& pair : faceProximities)
        if (pair.second.first < 2.0f * magic->repulsionThickness)
            ans.emplace_back(pair.second.second, pair.first.first, magic->collisionStiffness, clothFriction, obstacleFriction);
    
    return ans;
}

void Simulator::updateActive(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, const std::vector<Impact>& impacts) const {
    for (BVH* clothBvh : clothBvhs)
        clothBvh->setAllActive(false);
    for (BVH* obstacleBvh : obstacleBvhs)
        obstacleBvh->setAllActive(false);
    
    for (const Impact& impact : impacts)
        for (int i = 0; i < 4; i++) {
            Node* node = impact.nodes[i];
            for (BVH* clothBvh : clothBvhs)
                if (clothBvh->contain(node))
                    clothBvh->setActive(node, true);
            for (BVH* obstacleBvh : obstacleBvhs)
                if (obstacleBvh->contain(node))
                    obstacleBvh->setActive(node, true);
        }
}

void Simulator::checkImpacts(const Face* face0, const Face* face1, float thickness, std::vector<Impact>& impacts) const {
    Impact impact;
    for (int i = 0; i < 3; i++)
        if (checkVertexFaceImpact(face0->vertices[i], face1, thickness, impact))
            impacts.push_back(impact);
    for (int i = 0; i < 3; i++)
        if (checkVertexFaceImpact(face1->vertices[i], face0, thickness, impact))
            impacts.push_back(impact);
    for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
            if (checkEdgeEdgeImpact(face0->edges[i], face1->edges[j], thickness, impact))
                impacts.push_back(impact);
}

thrust::device_vector<Impact> Simulator::findImpacts(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs) const {
    thrust::device_vector<PairFF> pairs = std::move(traverse(clothBvhs, obstacleBvhs, magic->collisionThickness));
    int nPairs = pairs.size();
    thrust::device_vector<Impact> ans(15 * nPairs);
    checkImpactsGpu<<<GRID_SIZE, BLOCK_SIZE>>>(nPairs, pointer(pairs), magic->collisionThickness, pointer(ans));
    CUDA_CHECK_LAST();

    ans.erase(thrust::remove_if(ans.begin(), ans.end(), IsNull()), ans.end());
    return ans;
}

std::vector<Impact> Simulator::independentImpacts(const std::vector<Impact>& impacts, int deform) const {
    std::vector<Impact> sorted = impacts;
    std::sort(sorted.begin(), sorted.end());
    
    std::unordered_set<Node*> nodes;
    std::vector<Impact> ans;
    for (const Impact& impact : sorted) {
        bool flag = true;
        for (int i = 0; i < 4; i++) {
            Node* node = impact.nodes[i];
            if ((deform == 1 || node->isFree) && nodes.find(node) != nodes.end()) {
                flag = false;
                break;
            }
        }

        if (flag) {
            ans.push_back(impact);
            for (int i = 0; i < 4; i++)
                nodes.insert(impact.nodes[i]);
        }
    }
    return ans;
}

thrust::device_vector<Impact> Simulator::independentImpacts(const thrust::device_vector<Impact>& impacts, int deform) const {
    int nImpacts = impacts.size();
    const Impact* impactsPointer = pointer(impacts);
    int nNodes = 4 * nImpacts;
    thrust::device_vector<Node*> nodes(nNodes), outputNodes(nNodes);
    Node** nodesPointer = pointer(nodes);
    Node** outputNodesPointer = pointer(outputNodes);
    thrust::device_vector<Pairfi> relativeImpacts(nNodes), outputRelativeImpacts(nNodes);
    Pairfi* relativeImpactsPointer = pointer(relativeImpacts);
    Pairfi* outputRelativeImpactsPointer = pointer(outputRelativeImpacts);
    thrust::device_vector<Impact> ans(nImpacts);
    Impact* ansPointer = pointer(ans);
    initializeImpactNodes<<<GRID_SIZE, BLOCK_SIZE>>>(nImpacts, impactsPointer, deform);
    CUDA_CHECK_LAST();

    int num, newNum = nImpacts;
    do {
        num = newNum;
        collectRelativeImpacts<<<GRID_SIZE, BLOCK_SIZE>>>(nImpacts, impactsPointer, deform, nodesPointer, relativeImpactsPointer);
        CUDA_CHECK_LAST();

        thrust::sort_by_key(nodes.begin(), nodes.end(), relativeImpacts.begin());
        auto iter = thrust::reduce_by_key(nodes.begin(), nodes.end(), relativeImpacts.begin(), outputNodes.begin(), outputRelativeImpacts.begin(), thrust::equal_to<Node*>(), thrust::minimum<Pairfi>());

        setImpactMinIndices<<<GRID_SIZE, BLOCK_SIZE>>>(iter.first - outputNodes.begin(), outputRelativeImpactsPointer, outputNodesPointer);
        CUDA_CHECK_LAST();

        checkIndependentImpacts<<<GRID_SIZE, BLOCK_SIZE>>>(nImpacts, impactsPointer, deform, ansPointer);
        CUDA_CHECK_LAST();

        newNum = thrust::count_if(ans.begin(), ans.end(), IsNull());
    } while (num > newNum);

    ans.erase(thrust::remove_if(ans.begin(), ans.end(), IsNull()), ans.end());
    CUDA_CHECK_LAST();

    return ans;
}

void Simulator::updateActive(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, const std::vector<Intersection>& intersections) const {
    for (BVH* clothBvh : clothBvhs)
        clothBvh->setAllActive(false);
    for (BVH* obstacleBvh : obstacleBvhs)
        obstacleBvh->setAllActive(false);
    
    for (const Intersection& intersection : intersections)
        for (int i = 0; i < 3; i++) {
            Node* node0 = intersection.face0->vertices[i]->node;
            for (BVH* clothBvh : clothBvhs)
                if (clothBvh->contain(node0))
                    clothBvh->setActive(node0, true);
            for (BVH* obstacleBvh : obstacleBvhs)
                if (obstacleBvh->contain(node0))
                    obstacleBvh->setActive(node0, true);
            
            Node* node1 = intersection.face1->vertices[i]->node;
            for (BVH* clothBvh : clothBvhs)
                if (clothBvh->contain(node1))
                    clothBvh->setActive(node1, true);
            for (BVH* obstacleBvh : obstacleBvhs)
                if (obstacleBvh->contain(node1))
                    obstacleBvh->setActive(node1, true);
        }
}

Vector3f Simulator::oldPosition(const Vector2f& u, const std::vector<BackupFace>& faces) const {
    for (const BackupFace& face : faces) {
        Vector3f b = face.barycentricCoordinates(u);
        if (b(0) >= -1e-6f && b(1) >= -1e-6f && b(2) >= -1e-5f)
            return face.position(b);
    }
}

Vector3f Simulator::oldPosition(const Face* face, const Vector3f& b, const std::vector<std::vector<BackupFace>>& faces) const {
    if (!face->isFree())
        return face->position(b);
    
    Vector2f u = b(0) * face->vertices[0]->u + b(1) * face->vertices[1]->u + b(2) * face->vertices[2]->u;
    for (int i = 0; i < cloths.size(); i++)
        if (cloths[i]->getMesh()->contain(face))
            return oldPosition(u, faces[i]);
}

void Simulator::checkIntersection(const Face* face0, const Face* face1, std::vector<Intersection>& intersections, const std::vector<std::vector<BackupFace>>& faces) const {
    Intersection intersection;
    Vector3f& b0 = intersection.b0;
    Vector3f& b1 = intersection.b1;
    if (checkIntersectionMidpoint(face0, face1, b0, b1)) {
        intersection.face0 = const_cast<Face*>(face0);
        intersection.face1 = const_cast<Face*>(face1);
        Vector3f x0 = oldPosition(face0, b0, faces);
        Vector3f x1 = oldPosition(face1, b1, faces);
        Vector3f& d = intersection.d;
        d = (x0 - x1).normalized();
        farthestPoint(face0, face1, d, b0, b1);
        intersections.push_back(intersection);
    }
}

thrust::device_vector<Intersection> Simulator::findIntersections(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, const std::vector<thrust::device_vector<BackupFace>>& faces) const {
    thrust::device_vector<PairFF> pairs = std::move(traverse(clothBvhs, obstacleBvhs, magic->collisionThickness));
    int nPairs = pairs.size();
    thrust::device_vector<Intersection> ans(nPairs);
    Intersection* ansPointer = pointer(ans);
    checkIntersectionsGpu<<<GRID_SIZE, BLOCK_SIZE>>>(nPairs, pointer(pairs), ansPointer);
    CUDA_CHECK_LAST();

    ans.erase(thrust::remove_if(ans.begin(), ans.end(), IsNull()), ans.end());
    int nAns = ans.size();
    thrust::device_vector<Vector3f> x(2 * nAns);
    Vector3f* xPointer = pointer(x);
    initializeOldPosition<<<GRID_SIZE, BLOCK_SIZE>>>(nAns, ansPointer, xPointer);
    CUDA_CHECK_LAST();

    for (int i = 0; i < cloths.size(); i++) {
        thrust::device_vector<Vertex*>& vertices = cloths[i]->getMesh()->getVerticesGpu();
        thrust::device_vector<int> indices(2 * nAns);
        int* indicesPointer = pointer(indices);
        thrust::device_vector<Vector2f> u(2 * nAns);
        Vector2f* uPointer = pointer(u);
        collectContainedFaces<<<GRID_SIZE, BLOCK_SIZE>>>(nAns, ansPointer, vertices.size(), pointer(vertices), indicesPointer, uPointer);
        CUDA_CHECK_LAST();

        u.erase(thrust::remove_if(u.begin(), u.end(), indices.begin(), IsNull()), u.end());
        indices.erase(thrust::remove(indices.begin(), indices.end(), -1), indices.end());
        computeOldPosition<<<GRID_SIZE, BLOCK_SIZE>>>(indices.size(), indicesPointer, uPointer, faces[i].size(), pointer(faces[i]), xPointer);
        CUDA_CHECK_LAST();
    }

    computeFarthestPoint<<<GRID_SIZE, BLOCK_SIZE>>>(nAns, xPointer, ansPointer);
    CUDA_CHECK_LAST();

    return ans;
}

void Simulator::physicsStep() {
    for (Obstacle* obstacle : obstacles)
        obstacle->step(nSteps * dt, dt);

    std::vector<BVH*> clothBvhs = std::move(buildClothBvhs(false));
    std::vector<BVH*> obstacleBvhs = std::move(buildObstacleBvhs(false));

    if (!gpu) {
        std::vector<Proximity> proximities = std::move(findProximities(clothBvhs, obstacleBvhs));
        for (Cloth* cloth : cloths)
            cloth->physicsStep(dt, gravity, wind, magic->handleStiffness, proximities, magic->repulsionThickness);
    } else {
        for (Cloth* cloth : cloths)
            cloth->physicsStep(dt, gravity, wind, magic->handleStiffness);
    }

    destroyBvhs(clothBvhs);
    destroyBvhs(obstacleBvhs);

    for (Cloth* cloth : cloths)
        cloth->getMesh()->updatePositions(dt);
    for (Obstacle* obstacle : obstacles)
        obstacle->getMesh()->updatePositions(dt);
    
    updateClothFaceGeometries();
    updateClothNodeGeometries();
    updateObstacleFaceGeometries();
    updateObstacleNodeGeometries();
}

void Simulator::collisionStep() {
    std::vector<BVH*> clothBvhs = std::move(buildClothBvhs(true));
    std::vector<BVH*> obstacleBvhs = std::move(buildObstacleBvhs(true));
    int deform;
    float obstacleMass = 1e3f;

    if (!gpu) {
        std::vector<Impact> impacts;
        for (deform = 0; deform < 2; deform++) {
            impacts.clear();
            bool success = false;
            for (int i = 0; i < MAX_COLLISION_ITERATION; i++) {
                if (!impacts.empty())
                    updateActive(clothBvhs, obstacleBvhs, impacts);
                
                std::vector<Impact> newImpacts;
                traverse(clothBvhs, obstacleBvhs, magic->collisionThickness, [&](const Face* face0, const Face* face1, float thickness) {
                    checkImpacts(face0, face1, thickness, newImpacts);
                });
                if (newImpacts.empty()) {
                    success = true;
                    break;
                }

                newImpacts = std::move(independentImpacts(newImpacts, deform));
                impacts.insert(impacts.end(), newImpacts.begin(), newImpacts.end());
                Optimization* optimization = new CollisionOptimization(impacts, magic->collisionThickness, deform, obstacleMass);
                optimization->solve();
                delete optimization;

                updateBvhs(clothBvhs);
                if (deform == 1) {
                    updateBvhs(obstacleBvhs);
                    obstacleMass *= 0.5f;
                }
            }
            if (success)
                break;
        }
    } else {
        thrust::device_vector<Impact> impacts;
        for (deform = 0; deform < 2; deform++) {
            impacts.clear();
            bool success = false;
            for (int i = 0; i < MAX_COLLISION_ITERATION; i++) {
                thrust::device_vector<Impact> newImpacts = std::move(findImpacts(clothBvhs, obstacleBvhs));
                if (newImpacts.empty()) {
                    success = true;
                    break;
                }

                newImpacts = std::move(independentImpacts(newImpacts, deform));
                impacts.insert(impacts.end(), newImpacts.begin(), newImpacts.end());
                Optimization* optimization = new CollisionOptimization(impacts, magic->collisionThickness, deform, obstacleMass);
                optimization->solve();
                delete optimization;

                updateBvhs(clothBvhs);
                if (deform == 1) {
                    updateBvhs(obstacleBvhs);
                    obstacleMass *= 0.5f;
                }
            }
            if (success)
                break;
        }
    }

    destroyBvhs(clothBvhs);
    destroyBvhs(obstacleBvhs);

    updateClothFaceGeometries();
    updateClothNodeGeometries();
    updateClothVelocities();
    if (deform == 1) {
        updateObstacleFaceGeometries();
        updateObstacleNodeGeometries();
        updateObstacleVelocities();
    }
}

void Simulator::remeshingStep() {
    std::vector<BVH*> obstacleBvhs = std::move(buildObstacleBvhs(false));
    for (Cloth* cloth : cloths)
        cloth->remeshingStep(obstacleBvhs, 10.0f * magic->repulsionThickness);

    destroyBvhs(obstacleBvhs);

    updateClothFaceGeometries();
    updateClothNodeGeometries();
}

void Simulator::separationStep(const std::vector<std::vector<BackupFace>>& faces) {
    std::vector<BVH*> clothBvhs = std::move(buildClothBvhs(false));
    std::vector<BVH*> obstacleBvhs = std::move(buildObstacleBvhs(false));
    int deform;
    float obstacleArea = 1e3f;

    std::vector<Intersection> intersections;
    for (deform = 0; deform < 2; deform++) {
        intersections.clear();
        bool success = false;
        for (int i = 0; i < MAX_SEPARATION_ITERATION; i++) {
            if (!intersections.empty())
                updateActive(clothBvhs, obstacleBvhs, intersections);
            
            std::vector<Intersection> newIntersections;
            traverse(clothBvhs, obstacleBvhs, magic->collisionThickness, [&](const Face* face0, const Face* face1, float thickness) {
                checkIntersection(face0, face1, newIntersections, faces);
            });
            if (newIntersections.empty()) {
                success = true;
                break;
            }

            intersections.insert(intersections.end(), newIntersections.begin(), newIntersections.end());
            Optimization* optimization = new SeparationOptimization(intersections, magic->collisionThickness, deform, obstacleArea);
            optimization->solve();
            delete optimization;

            updateClothFaceGeometries();
            updateBvhs(clothBvhs);
            if (deform == 1) {
                updateObstacleFaceGeometries();
                updateBvhs(obstacleBvhs);
                obstacleArea *= 0.5f;
            }
        }
        if (success)
            break;
    }

    destroyBvhs(clothBvhs);
    destroyBvhs(obstacleBvhs);

    updateClothNodeGeometries();
    updateClothVelocities();
    if (deform == 1) {
        updateObstacleNodeGeometries();
        updateObstacleVelocities();
    }
}

void Simulator::separationStep(const std::vector<thrust::device_vector<BackupFace>>& faces) {
    std::vector<BVH*> clothBvhs = std::move(buildClothBvhs(false));
    std::vector<BVH*> obstacleBvhs = std::move(buildObstacleBvhs(false));
    int deform;
    float obstacleArea = 1e3f;

    thrust::device_vector<Intersection> intersections;
    for (deform = 0; deform < 2; deform++) {
        intersections.clear();
        bool success = false;
        for (int i = 0; i < MAX_SEPARATION_ITERATION; i++) {
            thrust::device_vector<Intersection> newIntersections = std::move(findIntersections(clothBvhs, obstacleBvhs, faces));
            if (newIntersections.empty()) {
                success = true;
                break;
            }

            intersections.insert(intersections.end(), newIntersections.begin(), newIntersections.end());
            Optimization* optimization = new SeparationOptimization(intersections, magic->collisionThickness, deform, obstacleArea);
            optimization->solve();
            delete optimization;

            updateClothFaceGeometries();
            updateBvhs(clothBvhs);
            if (deform == 1) {
                updateObstacleFaceGeometries();
                updateBvhs(obstacleBvhs);
                obstacleArea *= 0.5f;
            }
        }
        if (success)
            break;
    }

    destroyBvhs(clothBvhs);
    destroyBvhs(obstacleBvhs);

    updateClothNodeGeometries();
    updateClothVelocities();
    if (deform == 1) {
        updateObstacleNodeGeometries();
        updateObstacleVelocities();
    }
}

void Simulator::updateClothNodeGeometries() {
    for (Cloth* cloth : cloths)
        cloth->getMesh()->updateNodeGeometries();
}

void Simulator::updateObstacleNodeGeometries() {
    for (Obstacle* obstacle : obstacles)
        obstacle->getMesh()->updateNodeGeometries();
}

void Simulator::updateClothFaceGeometries() {
    for (Cloth* cloth : cloths)
        cloth->getMesh()->updateFaceGeometries();
}

void Simulator::updateObstacleFaceGeometries() {
    for (Obstacle* obstacle : obstacles)
        obstacle->getMesh()->updateFaceGeometries();
}

void Simulator::updateClothVelocities() {
    for (Cloth* cloth : cloths)
        cloth->getMesh()->updateVelocities(dt);
}

void Simulator::updateObstacleVelocities() {
    for (Obstacle* obstacle : obstacles)
        obstacle->getMesh()->updateVelocities(dt);
}

void Simulator::updateRenderingData(bool rebind) {
    for (Cloth* cloth : cloths)
        cloth->getMesh()->updateRenderingData(rebind);
    for (Obstacle* obstacle : obstacles)
        obstacle->getMesh()->updateRenderingData(false);
}

void Simulator::simulateStep(bool offline) {
    if ((++nSteps) % frameSteps == 0)
        nFrames++;
    std::cout << "Frame [" << nFrames << "], Step [" << nSteps << "]:" << std::endl;

    std::chrono::duration<float> d;
    auto t0 = std::chrono::high_resolution_clock::now();
    
    physicsStep();
    auto t1 = std::chrono::high_resolution_clock::now();
    d = t1 - t0;
    std::cout << "Physics Step: " << d.count() << "s";
    
    collisionStep();
    auto t2 = std::chrono::high_resolution_clock::now();
    d = t2 - t1;
    std::cout << ", Collision Step: " << d.count() << "s";
    
    if (nSteps % frameSteps == 0) {
        if (!gpu) {
            std::vector<std::vector<BackupFace>> faces(cloths.size());
            for (int i = 0; i < cloths.size(); i++)
                faces[i] = std::move(cloths[i]->getMesh()->backupFaces());

            remeshingStep();
            auto t3 = std::chrono::high_resolution_clock::now();
            d = t3 - t2;
            std::cout << ", Remeshing Step: " << d.count() << "s";

            separationStep(faces);
            auto t4 = std::chrono::high_resolution_clock::now();
            d = t4 - t3;
            std::cout << ", Separation Step: " << d.count() << "s";
        } else {
            std::vector<thrust::device_vector<BackupFace>> faces(cloths.size());
            for (int i = 0; i < cloths.size(); i++)
                faces[i] = std::move(cloths[i]->getMesh()->backupFacesGpu());

            remeshingStep();
            auto t3 = std::chrono::high_resolution_clock::now();
            d = t3 - t2;
            std::cout << ", Remeshing Step: " << d.count() << "s";

            separationStep(faces);
            auto t4 = std::chrono::high_resolution_clock::now();
            d = t4 - t3;
            std::cout << ", Separation Step: " << d.count() << "s";
        }

        if (!offline)
            updateRenderingData(true);
        else
            save();
    } else if (!offline)
        updateRenderingData(false);

    std::cout << std::endl;
}

void Simulator::replayStep() {
    if (nFrames == endFrame)
        return;

    nSteps += frameSteps;
    nFrames++;
    std::cout << "Frame [" << nFrames << "]" << std::endl;

    if (load()) {
        for (Obstacle* obstacle : obstacles)
            obstacle->transform(nSteps * dt);
        updateRenderingData(true);
    }
}

void Simulator::bind() {
    for (Cloth* cloth : cloths)
        cloth->bind();
    for (Obstacle* obstacle : obstacles)
        obstacle->bind();
}

void Simulator::render() const {
    Vector3f lightDirection = renderer->getLightDirection();
    Vector3f cameraPosition = renderer->getCameraPosition();
    Matrix4x4f model = renderer->getModel();
    Matrix4x4f view = renderer->getView();
    Matrix4x4f projection = renderer->getProjection();

    for (const Cloth* cloth : cloths)
        cloth->render(model, view, projection, cameraPosition, lightDirection);
    
    for (const Obstacle* obstacle : obstacles)
        obstacle->render(model, view, projection, cameraPosition, lightDirection);
}

bool Simulator::load() {
    for (int i = 0; i < cloths.size(); i++) {
        std::string path = directory + "/frame" + std::to_string(nFrames) + "_cloth" + std::to_string(i) + ".obj";
        if (!std::filesystem::exists(path))
            return false;
    }

    for (int i = 0; i < cloths.size(); i++) {
        std::string path = directory + "/frame" + std::to_string(nFrames) + "_cloth" + std::to_string(i) + ".obj";
        cloths[i]->load(path);
    }
}

void Simulator::save() {
    for (int i = 0; i < cloths.size(); i++)
        cloths[i]->save(directory + "/frame" + std::to_string(nFrames) + "_cloth" + std::to_string(i) + ".obj", json["cloths"][i]);

    std::ofstream fout(directory + "/config.json");
    fout << json;
}

int Simulator::lastFrame() const {
    int ans = 0;
    bool flag;
    do {
        ans++;
        flag = true;
        for (int i = 0; i < cloths.size(); i++)
            if (!std::filesystem::exists(directory + "/frame" + std::to_string(ans) + "_cloth" + std::to_string(i) + ".obj")) {
                flag = false;
                break;
            }
    } while (flag);

    return ans - 1;
}

bool Simulator::finished() const {
    return nSteps * dt >= endTime || nFrames >= endFrame;
}

void Simulator::simulate() {
    if (!gpu) {
        std::vector<std::vector<BackupFace>> faces(cloths.size());
        for (int i = 0; i < cloths.size(); i++)
            faces[i] = std::move(cloths[i]->getMesh()->backupFaces());

        remeshingStep();
        separationStep(faces);
    } else {
        std::vector<thrust::device_vector<BackupFace>> faces(cloths.size());
        for (int i = 0; i < cloths.size(); i++)
            faces[i] = std::move(cloths[i]->getMesh()->backupFacesGpu());

        remeshingStep();
        separationStep(faces);
    }
    bind();

    while (!glfwWindowShouldClose(renderer->getWindow()) && !finished()) {
        glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        render();
        if (!renderer->getPause())
            simulateStep(false);                

        glfwSwapBuffers(renderer->getWindow());
        glfwPollEvents();
    }
}

void Simulator::simulateOffline() {
    if (!std::filesystem::is_directory(directory))
        std::filesystem::create_directories(directory);

    if (!gpu) {
        std::vector<std::vector<BackupFace>> faces(cloths.size());
        for (int i = 0; i < cloths.size(); i++)
            faces[i] = std::move(cloths[i]->getMesh()->backupFaces());

        remeshingStep();
        separationStep(faces);
    } else {
        std::vector<thrust::device_vector<BackupFace>> faces(cloths.size());
        for (int i = 0; i < cloths.size(); i++)
            faces[i] = std::move(cloths[i]->getMesh()->backupFacesGpu());

        remeshingStep();
        separationStep(faces);
    }
    save();

    while (!finished())
        simulateStep(true);
}

void Simulator::resume() {
    // nFrames = lastFrame();
    nFrames = 57;
    nSteps = nFrames * frameSteps;
    for (Obstacle* obstacle : obstacles)
        obstacle->transform(nSteps * dt);
    bind();
    
    while (!glfwWindowShouldClose(renderer->getWindow()) && !finished()) {
        glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        render();
        if (!renderer->getPause())
            simulateStep(false);

        glfwSwapBuffers(renderer->getWindow());
        glfwPollEvents();
    }
}

void Simulator::resumeOffline() {
    nFrames = lastFrame();
    nSteps = nFrames * frameSteps;
    for (Obstacle* obstacle : obstacles)
        obstacle->transform(nSteps * dt);

    while (!finished())
        simulateStep(true);
}

void Simulator::replay() {
    endFrame = lastFrame();
    load();
    bind();

    while (!glfwWindowShouldClose(renderer->getWindow())) {
        auto t0 = std::chrono::high_resolution_clock::now();
       
        glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        render();
        if (!renderer->getPause())
            replayStep();

        glfwSwapBuffers(renderer->getWindow());
        glfwPollEvents();

        std::chrono::duration<float> d;
        do {
            auto t1 = std::chrono::high_resolution_clock::now();
            d = t1 - t0;
        } while (d.count() < frameTime);
    }
}

void Simulator::start() {
    switch (mode) {
    case Simulate:
        simulate();
        break;
    case SimulateOffline:
        simulateOffline();
        break;
    case Resume:
        resume();
        break;
    case ResumeOffline:
        resumeOffline();
        break;
    case Replay:
        replay();
        break;
    }
}
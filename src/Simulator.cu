#include "Simulator.cuh"

Simulator::Simulator(const std::string& path) :
    nSteps(0),
    selectedCloth(-1) {
    std::ifstream fin(path);
    if (!fin.is_open()) {
        std::cerr << "Failed to open configuration file: " << path << std::endl;
        exit(1);
    }

    Json::Value json;
    fin >> json;

    frameTime = parseFloat(json["frame_time"]);
    frameSteps = parseInt(json["frame_steps"]);
    dt =  frameTime / frameSteps;

    gravity = parseVector3f(json["gravity"]);
    Wind* windTemp = new Wind();

    magic = new Magic(json["magic"]);
    
    cloths.resize(json["cloths"].size());
    for (int i = 0; i < json["cloths"].size(); i++)
        cloths[i] = new Cloth(json["cloths"][i]);

    obstacles.resize(json["obstacles"].size());
    for (int i = 0; i < json["obstacles"].size(); i++)
        obstacles[i] = new Obstacle(json["obstacles"][i]);

    fin.close();

    // cloths[0]->readDataFromFile("input.txt");
    remeshingStep();
    bind();

    glGenFramebuffers(1, &fbo);
    glBindFramebuffer(GL_FRAMEBUFFER, fbo);
    glGenTextures(1, &indexTexture);
    glGenRenderbuffers(1, &rbo);
    glBindFramebuffer(GL_FRAMEBUFFER, 0);

    indexShader = new Shader("shader/Vertex.glsl", "shader/IndexFragment.glsl");

    if (!gpu)
        wind = windTemp;
    else {
        CUDA_CHECK(hipMalloc(&wind, sizeof(Wind)));
        CUDA_CHECK(hipMemcpy(wind, windTemp, sizeof(Wind), hipMemcpyHostToDevice));
        delete windTemp;
    }
}

Simulator::~Simulator() {
    delete magic;
    for (const Cloth* cloth : cloths)
        delete cloth;
    for (const Obstacle* obstacle : obstacles)
        delete obstacle;
    delete indexShader;

    if (!gpu)
        delete wind;
    else
        CUDA_CHECK(hipFree(wind));
}

std::vector<BVH*> Simulator::buildClothBvhs(bool ccd) const {
    std::vector<BVH*> ans(cloths.size());
    for (int i = 0; i < cloths.size(); i++)
        ans[i] = new BVH(cloths[i]->getMesh(), ccd);
    return ans;
}

std::vector<BVH*> Simulator::buildObstacleBvhs(bool ccd) const {
    std::vector<BVH*> ans(obstacles.size());
    for (int i = 0; i < obstacles.size(); i++)
        ans[i] = new BVH(obstacles[i]->getMesh(), ccd);
    return ans;
}

void Simulator::updateBvhs(std::vector<BVH*>& bvhs) const {
    for (BVH* bvh : bvhs)
        bvh->update();
}

void Simulator::destroyBvhs(const std::vector<BVH*>& bvhs) const {
    for (const BVH* bvh : bvhs)
        delete bvh;
}

void Simulator::traverse(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, float thickness, std::function<void(const Face*, const Face*, float)> callback) const {
    for (int i = 0; i < clothBvhs.size(); i++) {
        clothBvhs[i]->traverse(thickness, callback);
        for (int j = 0; j < i; j++)
            clothBvhs[i]->traverse(clothBvhs[j], thickness, callback);
        
        for (int j = 0; j < obstacleBvhs.size(); j++)
            clothBvhs[i]->traverse(obstacleBvhs[j], thickness, callback);
    }
}

std::vector<Impact> Simulator::independentImpacts(const std::vector<Impact>& impacts) const {
    std::vector<Impact> sorted = impacts;
    std::sort(sorted.begin(), sorted.end());
    
    std::unordered_set<Node*> nodes;
    std::vector<Impact> ans;
    for (const Impact& impact : sorted) {
        bool flag = true;
        for (int i = 0; i < 4; i++)
            if (impact.nodes[i]->isFree && nodes.find(impact.nodes[i]) != nodes.end()) {
                flag = false;
                break;
            }
        if (flag) {
            ans.push_back(impact);
            for (int i = 0; i < 4; i++)
                nodes.insert(impact.nodes[i]);
        }
    }
    return ans;
}

void Simulator::updateActive(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, const std::vector<Impact>& impacts) const {
    for (BVH* clothBvh : clothBvhs)
        clothBvh->setAllActive(false);
    for (BVH* obstacleBvh : obstacleBvhs)
        obstacleBvh->setAllActive(false);
    
    for (const Impact& impact : impacts)
        for (int i = 0; i < 4; i++) {
            Node* node = impact.nodes[i];
            for (BVH* clothBvh : clothBvhs)
                if (clothBvh->contain(node))
                    clothBvh->setActive(node, true);
            for (BVH* obstacleBvh : obstacleBvhs)
                if (obstacleBvh->contain(node))
                    obstacleBvh->setActive(node, true);
        }
}

void Simulator::updateActive(const std::vector<BVH*>& clothBvhs, const std::vector<BVH*>& obstacleBvhs, const std::vector<Intersection>& intersections) const {
    for (BVH* clothBvh : clothBvhs)
        clothBvh->setAllActive(false);
    for (BVH* obstacleBvh : obstacleBvhs)
        obstacleBvh->setAllActive(false);
    
    for (const Intersection& intersection : intersections)
        for (int i = 0; i < 3; i++) {
            Node* node0 = intersection.face0->vertices[i]->node;
            for (BVH* clothBvh : clothBvhs)
                if (clothBvh->contain(node0))
                    clothBvh->setActive(node0, true);
            for (BVH* obstacleBvh : obstacleBvhs)
                if (obstacleBvh->contain(node0))
                    obstacleBvh->setActive(node0, true);
            
            Node* node1 = intersection.face1->vertices[i]->node;
            for (BVH* clothBvh : clothBvhs)
                if (clothBvh->contain(node1))
                    clothBvh->setActive(node1, true);
            for (BVH* obstacleBvh : obstacleBvhs)
                if (obstacleBvh->contain(node1))
                    obstacleBvh->setActive(node1, true);
        }
}

void Simulator::resetObstacles() {
    for (Obstacle* obstacle : obstacles)
        obstacle->reset();
}

void Simulator::physicsStep() {
    for (Cloth* cloth : cloths)
        cloth->physicsStep(dt, magic->handleStiffness, gravity, wind);
    updateGeometries();
}

void Simulator::collisionStep() {
    std::vector<BVH*> clothBvhs = std::move(buildClothBvhs(true));
    std::vector<BVH*> obstacleBvhs = std::move(buildObstacleBvhs(true));
    float obstacleMass = 1e3f;

    if (!gpu) {
        std::vector<Impact> impacts;
        for (int deform = 0; deform < 2; deform++) {
            impacts.clear();
            bool success = false;
            for (int i = 0; i < MAX_COLLISION_ITERATION; i++) {
                if (!impacts.empty())
                    updateActive(clothBvhs, obstacleBvhs, impacts);
                
                std::vector<Impact> newImpacts;
                traverse(clothBvhs, obstacleBvhs, magic->collisionThickness, [&](const Face* face0, const Face* face1, float thickness) {
                    checkImpacts(face0, face1, thickness, newImpacts);
                });
                newImpacts = std::move(independentImpacts(newImpacts));
                if (newImpacts.empty()) {
                    success = true;
                    break;
                }

                impacts.insert(impacts.end(), newImpacts.begin(), newImpacts.end());
                Optimization* optimization = new CollisionOptimization(impacts, magic->collisionThickness, deform, obstacleMass);
                augmentedLagrangianMethod(optimization);
                delete optimization;

                updateBvhs(clothBvhs);
                if (deform == 1) {
                    updateBvhs(obstacleBvhs);
                    obstacleMass *= 0.5f;
                }
            }
            if (success)
                break;
        }
    } else {
        // TODO
    }

    destroyBvhs(clothBvhs);
    destroyBvhs(obstacleBvhs);

    updateGeometries();
    updateVelocities();
}

void Simulator::remeshingStep() {
    if (!gpu) {
        std::vector<BVH*> obstacleBvhs = std::move(buildObstacleBvhs(false));
        for (Cloth* cloth : cloths)
            cloth->remeshingStep(obstacleBvhs, 10.0f * magic->repulsionThickness);

        destroyBvhs(obstacleBvhs);
    } else {
        // TODO
    }

    updateStructures();
    updateGeometries();
}

void Simulator::separationStep(const std::vector<Mesh*>& oldMeshes) {
    if (!gpu) {
        std::vector<BVH*> clothBvhs = std::move(buildClothBvhs(false));
        std::vector<BVH*> obstacleBvhs = std::move(buildObstacleBvhs(false));
        std::vector<Intersection> intersections;
        
        for (int i = 0; i < MAX_SEPARATION_ITERATION; i++) {
            if (!intersections.empty())
                updateActive(clothBvhs, obstacleBvhs, intersections);
            
            std::vector<Intersection> newIntersections;
            traverse(clothBvhs, obstacleBvhs, magic->collisionThickness, [&](const Face* face0, const Face* face1, float thickness) {
                checkIntersection(face0, face1, newIntersections, cloths, oldMeshes);
            });
            if (newIntersections.empty())
                break;

            intersections.insert(intersections.end(), newIntersections.begin(), newIntersections.end());
            Optimization* optimization = new SeparationOptimization(intersections, magic->collisionThickness);
            augmentedLagrangianMethod(optimization);
            delete optimization;

            updateBvhs(clothBvhs);
        }

        destroyBvhs(clothBvhs);
        destroyBvhs(obstacleBvhs);
    } else {
        // TODO
    }

    updateGeometries();
    updateVelocities();
}

void Simulator::updateStructures() {
    for (Cloth* cloth : cloths)
        cloth->updateStructures();
}

void Simulator::updateGeometries() {
    for (Cloth* cloth : cloths)
        cloth->updateGeometries();
}

void Simulator::updateVelocities() {
    for (Cloth* cloth : cloths)
        cloth->updateVelocities(dt);
}

void Simulator::updateRenderingData(bool rebind) {
    for (Cloth* cloth : cloths)
        cloth->updateRenderingData(rebind);
}

void Simulator::bind() {
    for (Cloth* cloth : cloths)
        cloth->bind();
    for (Obstacle* obstacle : obstacles)
        obstacle->bind();
}

void Simulator::render(int width, int height, const Matrix4x4f& model, const Matrix4x4f& view, const Matrix4x4f& projection, const Vector3f& cameraPosition, const Vector3f& lightDirection) const {
    glBindFramebuffer(GL_DRAW_FRAMEBUFFER, fbo);
    glBindTexture(GL_TEXTURE_2D, indexTexture);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RG32I, width, height, 0, GL_RG_INTEGER, GL_INT, nullptr);
    glBindTexture(GL_TEXTURE_2D, 0);
    glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, indexTexture, 0);
    glBindRenderbuffer(GL_RENDERBUFFER, rbo);
    glRenderbufferStorage(GL_RENDERBUFFER, GL_DEPTH_COMPONENT32F, width, height);
    glBindRenderbuffer(GL_RENDERBUFFER, 0);
    glFramebufferRenderbuffer(GL_FRAMEBUFFER, GL_DEPTH_ATTACHMENT, GL_RENDERBUFFER, rbo);
    int color[2] = {-1, -1};
    glClearBufferiv(GL_COLOR, 0, color);
    glClear(GL_DEPTH_BUFFER_BIT);
    indexShader->use();
    indexShader->setMat4("model", model);
    indexShader->setMat4("view", view);
    indexShader->setMat4("projection", projection);
    for (int i = 0; i < cloths.size(); i++) {
        indexShader->setInt("clothIndex", i);
        cloths[i]->getMesh()->render();
    }
    glBindFramebuffer(GL_FRAMEBUFFER, 0);

    for (int i = 0; i < cloths.size(); i++)
        cloths[i]->render(model, view, projection, cameraPosition, lightDirection, selectedCloth == i ? selectedFace : -1);
    
    for (const Obstacle* obstacle : obstacles)
        obstacle->render(model, view, projection, cameraPosition, lightDirection);
}

void Simulator::step() {
    nSteps++;
    std::cout << "Step [" << nSteps << "]:" << std::endl;

    selectedCloth = -1;

    resetObstacles();
    
    std::chrono::duration<float> d;
    auto t0 = std::chrono::high_resolution_clock::now();
    
    physicsStep();
    auto t1 = std::chrono::high_resolution_clock::now();
    d = t1 - t0;
    std::cout << "Physics Step: " << d.count() << "s";
    
    collisionStep();
    auto t2 = std::chrono::high_resolution_clock::now();
    d = t2 - t1;
    std::cout << ", Collision Step: " << d.count() << "s";
    
    if (nSteps % frameSteps == 0) {
        std::vector<Mesh*> oldMeshes(cloths.size());
        for (int i = 0; i < cloths.size(); i++)
            oldMeshes[i] = new Mesh(cloths[i]->getMesh());

        remeshingStep();
        auto t3 = std::chrono::high_resolution_clock::now();
        d = t3 - t2;
        std::cout << ", Remeshing Step: " << d.count() << "s";

        separationStep(oldMeshes);
        auto t4 = std::chrono::high_resolution_clock::now();
        d = t4 - t3;
        std::cout << ", Separation Step: " << d.count() << "s";

        for (const Mesh* mesh : oldMeshes)
            delete mesh;

        updateRenderingData(true);
    } else
        updateRenderingData(false);

    std::cout << std::endl;
}

void Simulator::printDebugInfo(int x, int y) {
    glBindFramebuffer(GL_READ_FRAMEBUFFER, fbo);
    glReadBuffer(GL_COLOR_ATTACHMENT0);

    Pixel pixel;
    glReadPixels(x, y, 1, 1, GL_RG_INTEGER, GL_INT, &pixel);
    selectedCloth = pixel.clothIndex;
    selectedFace = pixel.faceInedx;
    if (selectedCloth != -1 && selectedFace != -1)
        cloths[selectedCloth]->printDebugInfo(selectedFace);

    glReadBuffer(GL_NONE);
    glBindFramebuffer(GL_FRAMEBUFFER, 0);
}

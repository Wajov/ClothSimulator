#include "hip/hip_runtime.h"
#include "RemeshingHelper.cuh"

__global__ void setX(int nNodes, const Node* const* nodes, Vector3f* x) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads)
        x[i] = nodes[i]->x;
}

__global__ void initializeNearPoints(int nNodes, const Vector3f* x, NearPoint* points) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads)
        points[i].x = x[i];
}

float unsignedVertexEdgeDistance(const Vector3f& x, const Vector3f& y0, const Vector3f& y1, Vector3f& n, float& wx, float& wy0, float& wy1) {
    float t = clamp((x - y0).dot(y1 - y0)/(y1 - y0).dot(y1 - y0), 0.0f, 1.0f);
    Vector3f y = y0 + t * (y1 - y0);
    float d = (x - y).norm();
    n = (x - y).normalized();
    wx = 1.0f;
    wy0 = 1.0f - t;
    wy1 = t;
    return d;
}

float unsignedVertexFaceDistance(const Vector3f& x, const Vector3f& y0, const Vector3f& y1, const Vector3f& y2, Vector3f& n, float* w) {
    Vector3f nt = (y1 - y0).cross(y2 - y0).normalized();
    float d = abs((x - y0).dot(nt));
    float b0 = mixed(y1 - x, y2 - x, nt);
    float b1 = mixed(y2 - x, y0 - x, nt);
    float b2 = mixed(y0 - x, y1 - x, nt);
    if (b0 >= 0.0f && b1 >= 0.0f && b2 >= 0.0f) {
        n = nt;
        w[0] = 1.0f;
        w[1] = -b0 / (b0 + b1 + b2);
        w[2] = -b1 / (b0 + b1 + b2);
        w[3] = -b2 / (b0 + b1 + b2);
        return d;
    }
    d = INFINITY;
    if (b0 < 0.0f) {
        float dt = unsignedVertexEdgeDistance(x, y1, y2, n, w[0], w[2], w[3]);
        if (dt < d) {
            d = dt;
            w[1] = 0.0f;
        }
    }
    if (b1 < 0.0f) {
        float dt = unsignedVertexEdgeDistance(x, y2, y0, n, w[0], w[3], w[1]);
        if (dt < d) {
            d = dt;
            w[2] = 0.0f;
        }
    }
    if (b2 < 0.0f) {
        float dt = unsignedVertexEdgeDistance(x, y0, y1, n, w[0], w[1], w[2]);
        if (dt < d) {
            d = dt;
            w[3] = 0.0f;
        }
    }
    return d;
}

void checkNearestPoint(const Vector3f& x, const Face* face, NearPoint& point) {
    Vector3f n;
    float w[4];
    Vector3f x1 = face->vertices[0]->node->x;
    Vector3f x2 = face->vertices[1]->node->x;
    Vector3f x3 = face->vertices[2]->node->x;
    float d = unsignedVertexFaceDistance(x, x1, x2, x3, n, w);

    if (d < point.d) {
        point.d = d;
        point.x = -(w[1] * x1 + w[2] * x2 + w[3] * x3);
    }
}

__global__ void setNearestPlane(int nNodes, const Vector3f* x, const NearPoint* points, Plane* planes) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads) {
        Vector3f xt = points[i].x;
        Vector3f n = x[i] - xt;
        if (n.norm2() > 1e-8f)
            planes[i] = Plane(xt, n.normalized());
    }
}

Matrix2x2f diagonal(const Vector2f& v) {
    Matrix2x2f ans;
    for (int i = 0; i < 2; i++)
        ans(i, i) = v(i);
    return ans;
}

Matrix2x2f sqrt(const Matrix2x2f& A) {
    Matrix2x2f Q;
    Vector2f l;
    eigenvalueDecomposition(A, Q, l);
    for (int i = 0; i < 2; i++)
        l(i) = l(i) >= 0.0f ? sqrt(l(i)) : -sqrt(-l(i));
    return Q * diagonal(l) * Q.transpose();
}

Matrix2x2f max(const Matrix2x2f& A, float v) {
    Matrix2x2f Q;
    Vector2f l;
    eigenvalueDecomposition(A, Q, l);
    for (int i = 0; i < 2; i++)
        l(i) = max(l(i), v);
    return Q * diagonal(l) * Q.transpose();
}

Matrix2x2f compressionMetric(const Matrix2x2f& G, const Matrix2x2f& S2, const Remeshing* remeshing) {
    Matrix2x2f P(Vector2f(S2(1, 1), -S2(1, 0)), Vector2f(-S2(0, 1), S2(0, 0)));
    Matrix2x2f D = G.transpose() * G - 4.0f * sqr(remeshing->refineCompression) * P * remeshing->ribStiffening;
    return max(-G + sqrt(D), 0.0f) / (2.0f * sqr(remeshing->refineCompression));
}

Matrix2x2f obstacleMetric(const Face* face, const Plane* planes) {
    Matrix2x2f ans;
    for (int i = 0; i < 3; i++) {
        Plane plane = planes[face->vertices[i]->node->index];
        if (plane.n.norm2() == 0.0f)
            continue;
        float h[3];
        for (int j = 0; j < 3; j++)
            h[j] = (face->vertices[j]->node->x - plane.p).dot(plane.n);
        Vector2f dh = face->inverse.transpose() * Vector2f(h[1] - h[0], h[2] - h[0]);
        ans += dh.outer(dh) / sqr(h[i]);
    }
    return ans / 3.0f;
}

Disk circumscribedDisk(const Disk& d0, const Disk& d1) {
    float d = (d0.o - d1.o).norm();
    float r = 0.5f * (d0.r + d + d1.r);
    float t = (r - d0.r) / d;
    return Disk(d0.o + t * (d1.o - d0.o), r);
}

Disk circumscribedDisk(const Disk& d0, const Disk& d1, const Disk& d2) {
    float x0 = d0.o(0), y0 = d0.o(1), r0 = d0.r;
    float x1 = d1.o(0), y1 = d1.o(1), r1 = d1.r;
    float x2 = d2.o(0), y2 = d2.o(1), r2 = d2.r;

    float v11 = 2.0f * x1 - 2.0f * x0;
    float v12 = 2.0f * y1 - 2.0f * y0;
    float v13 = sqr(x0) - sqr(x1) + sqr(y0) - sqr(y1) - sqr(r0) + sqr(r1);
    float v14 = 2.0f * r1 - 2.0f * r0;
    float v21 = 2.0f * x2 - 2.0f * x1;
    float v22 = 2.0f * y2 - 2.0f * y1;
    float v23 = sqr(x1) - sqr(x2) + sqr(y1) - sqr(y2) - sqr(r1) + sqr(r2);
    float v24 = 2.0f * r2 - 2.0f * r1;
    float w12 = v12 / v11;
    float w13 = v13 / v11;
    float w14 = v14 / v11;
    float w22 = v22 / v21 - w12;
    float w23 = v23 / v21 - w13;
    float w24 = v24 / v21 - w14;
    float P = -w23 / w22;
    float Q = w24 / w22;
    float M = - w12 * P - w13;
    float N = w14 - w12 * Q;
    float a = sqr(N) + sqr(Q) - 1.0f;
    float b = 2.0f * M * N - 2.0f * N * x0 + 2.0f * P * Q - 2.0f * Q * y0 + 2.0f * r0;
    float c = sqr(x0) + sqr(M) - 2.0f * M * x0 + sqr(P) + sqr(y0) - 2.0f * P * y0 - sqr(r0);
    float D = sqr(b) - 4.0f * a * c;
    float rs = (-b - sqrt(D)) / (2.0f * a);
    float xs = M + N * rs;
    float ys = P + Q * rs;

    return Disk(Vector2f(xs , ys), rs);
}

Matrix2x2f maxTensor(const Matrix2x2f* M) {
    int n = 0;
    Disk d[5];
    for (int i = 0; i < 5; i++)
        if (M[i].trace() != 0.0f) {
            d[n].o = Vector2f(0.5f * (M[i](0, 0) - M[i](1, 1)), 0.5f * (M[i](0, 1) + M[i](1, 0)));
            d[n].r = 0.5f * (M[i](0, 0) + M[i](1, 1));
            n++;
        }

    Disk disk;
    disk = d[0];
    for (int i = 1; i < n; i++)
        if (!disk.enclose(d[i])) {
            disk = d[i];
            for (int j = 0; j < i; j++)
                if (!disk.enclose(d[j])) {
                    disk = circumscribedDisk(d[i], d[j]);
                    for (int k = 0; k < j; k++)
                        if (!disk.enclose(d[k]))
                            disk = circumscribedDisk(d[i], d[j], d[k]);
                }
        }

    Matrix2x2f ans;
    ans(0, 0) = disk.r + disk.o(0);
    ans(0, 1) = ans(1, 0) = disk.o(1);
    ans(1, 1) = disk.r - disk.o(0);
    return ans;
}

Matrix2x2f faceSizing(const Face* face, const Plane* planes, const Remeshing* remeshing) {
    Node* node0 = face->vertices[0]->node;
    Node* node1 = face->vertices[1]->node;
    Node* node2 = face->vertices[2]->node;
    Matrix2x2f M[5];

    Matrix2x2f Sw1 = face->curvature();
    M[0] = (Sw1.transpose() * Sw1) / sqr(remeshing->refineAngle);
    Matrix3x2f Sw2 = face->derivative(node0->n, node1->n, node2->n);
    M[1] = (Sw2.transpose() * Sw2) / sqr(remeshing->refineAngle);
    Matrix3x2f V = face->derivative(node0->v, node1->v, node2->v);
    M[2] = (V.transpose() * V) / sqr(remeshing->refineVelocity);
    Matrix3x2f F = face->derivative(node0->x, node1->x, node2->x);
    M[3] = compressionMetric(F.transpose() * F - Matrix2x2f(1.0f), Sw2.transpose() * Sw2, remeshing);
    M[4] = obstacleMetric(face, planes);
    Matrix2x2f S = maxTensor(M);

    Matrix2x2f Q;
    Vector2f l;
    eigenvalueDecomposition(S, Q, l);
    for (int i = 0; i < 2; i++)
        l(i) = clamp(l(i), 1.0f / sqr(remeshing->sizeMax), 1.0f / sqr(remeshing->sizeMin));
    float lMax = max(l(0), l(1));
    float lMin = lMax * sqr(remeshing->aspectMin);
    for (int i = 0; i < 2; i++)
        l(i) = max(l(i), lMin);
    return Q * diagonal(l) * Q.transpose();
}

__global__ void initializeSizing(int nVertices, Vertex** vertices) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nVertices; i += nThreads) {
        Vertex* vertex = vertices[i];
        vertex->area = 0.0f;
        vertex->sizing = Matrix2x2f();
    }
}

__global__ void computeSizingGpu(int nFaces, const Face* const* faces, const Plane* planes, const Remeshing* remeshing) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nFaces; i += nThreads) {
        const Face* face = faces[i];
        float area = face->area;
        Matrix2x2f sizing = faceSizing(face, planes, remeshing);
        for (int j = 0; j < 3; j++) {
            Vertex* vertex = face->vertices[j];
            atomicAdd(&vertex->area, area);
            for (int k = 0; k < 2; k++)
                for (int h = 0; h < 2; h++)
                    atomicAdd(&vertex->sizing(k, h), area * sizing(k, h));
        }
    }
}

__global__ void finalizeSizing(int nVertices, Vertex** vertices) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nVertices; i += nThreads) {
        Vertex* vertex = vertices[i];
        vertex->sizing /= vertex->area;
    }
}

bool shouldFlip(const Edge* edge, const Remeshing* remeshing) {
    if (edge->isBoundary() || edge->isSeam())
        return false;
        
    Vertex* vertex0 = edge->vertices[0][0];
    Vertex* vertex1 = edge->vertices[1][1];
    Vertex* vertex2 = edge->opposites[0];
    Vertex* vertex3 = edge->opposites[1];

    Vector2f x = vertex0->u, y = vertex1->u, z = vertex2->u, w = vertex3->u;
    Matrix2x2f M = 0.25f * (vertex0->sizing + vertex1->sizing + vertex2->sizing + vertex3->sizing);
    float area0 = edge->adjacents[0]->area;
    float area1 = edge->adjacents[1]->area;
    return area1 * (x - z).dot(M * (y - z)) + area0 * (y - w).dot(M * (x - w)) < -remeshing->flipThreshold * (area0 + area1);
}

__global__ void checkEdgesToFlip(int nEdges, const Edge* const* edges, const Remeshing* remeshing, Edge** edgesToFlip) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        const Edge* edge = edges[i];
        edgesToFlip[i] = shouldFlip(edge, remeshing) ? const_cast<Edge*>(edge) : nullptr;
    }
}

__global__ void initializeEdgeNodes(int nEdges, const Edge* const* edges) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        const Edge* edge = edges[i];
        for (int j = 0; j < 2; j++)
            edge->nodes[j]->removed = false;
    }
}

__global__ void resetEdgeNodes(int nEdges, const Edge* const* edges) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        const Edge* edge = edges[i];
        Node* node0 = edge->nodes[0];
        Node* node1 = edge->nodes[1];
        if (!node0->removed && !node1->removed)
            node0->minIndex = node1->minIndex = nEdges;
    }
}

__global__ void computeEdgeMinIndices(int nEdges, const Edge* const* edges) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        const Edge* edge = edges[i];
        Node* node0 = edge->nodes[0];
        Node* node1 = edge->nodes[1];
        if (!node0->removed && !node1->removed) {
            atomicMin(&node0->minIndex, i);
            atomicMin(&node1->minIndex, i);
        }
    }
}

__global__ void checkIndependentEdges(int nEdges, const Edge* const* edges, Edge** independentEdges) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        const Edge* edge = edges[i];
        Node* node0 = edge->nodes[0];
        Node* node1 = edge->nodes[1];
        if (!node0->removed && !node1->removed && node0->minIndex == node1->minIndex) {
            independentEdges[i] = const_cast<Edge*>(edge);
            node0->removed = node1->removed = true;
        }
    }
}

__global__ void flipGpu(int nEdges, const Edge* const* edges, const Material* material, Edge** addedEdges, Edge** removedEdges, Face** addedFaces, Face** removedFaces) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        const Edge* edge = edges[i];
        Vertex* vertex0 = edge->vertices[0][0];
        Vertex* vertex1 = edge->vertices[1][1];
        Vertex* vertex2 = edge->opposites[0];
        Vertex* vertex3 = edge->opposites[1];

        Face* face0 = edge->adjacents[0];
        Face* face1 = edge->adjacents[1];

        Edge* edge0 = face0->findEdge(vertex1, vertex2);
        Edge* edge1 = face0->findEdge(vertex2, vertex0);
        Edge* edge2 = face1->findEdge(vertex0, vertex3);
        Edge* edge3 = face1->findEdge(vertex3, vertex1);

        Edge* newEdge = new Edge(vertex2->node, vertex3->node);
        Face* newFace0 = new Face(vertex0, vertex3, vertex2, material);
        Face* newFace1 = new Face(vertex1, vertex2, vertex3, material);
        newEdge->initialize(vertex0, newFace0);
        newEdge->initialize(vertex1, newFace1);
        newFace0->setEdges(edge2, newEdge, edge1);
        newFace1->setEdges(edge0, newEdge, edge3);

        edge0->initialize(vertex3, newFace1);
        edge1->initialize(vertex3, newFace0);
        edge2->initialize(vertex2, newFace0);
        edge3->initialize(vertex2, newFace1);

        addedEdges[i] = newEdge;
        removedEdges[i] = const_cast<Edge*>(edge);
        addedFaces[2 * i] = newFace0;
        addedFaces[2 * i + 1] = newFace1;
        removedFaces[2 * i] = face0;
        removedFaces[2 * i + 1] = face1;
    }
}

float edgeMetric(const Vertex* vertex0, const Vertex* vertex1) {
    if (vertex0 == nullptr || vertex1 == nullptr)
        return 0.0f;
    Vector2f du = vertex0->u - vertex1->u;
    return sqrt(0.5f * (du.dot(vertex0->sizing * du) + du.dot(vertex1->sizing * du)));
}

float edgeMetric(const Edge* edge) {
    return max(edgeMetric(edge->vertices[0][0], edge->vertices[0][1]), edgeMetric(edge->vertices[1][0], edge->vertices[1][1]));
}

__global__ void checkEdgesToSplit(int nEdges, const Edge* const* edges, Edge** edgesToSplit, float* metrics) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        const Edge* edge = edges[i];
        float m = edgeMetric(edge);
        if (m > 1.0f) {
            edgesToSplit[i] = const_cast<Edge*>(edge);
            metrics[i] = m;
        } else
            edgesToSplit[i] = nullptr;
    }
}

__global__ void splitGpu(int nEdges, const Edge* const* edges, const Material* material, Node** addedNodes, Vertex** addedVertices, Edge** addedEdges, Edge** removedEdges, Face** addedFaces, Face** removedFaces) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        const Edge* edge = edges[i];
        Node* node0 = edge->nodes[0];
        Node* node1 = edge->nodes[1];

        Node* newNode = new Node(0.5f * (node0->x + node1->x), node0->isFree && node1->isFree);
        newNode->x0 = 0.5f * (node0->x0 + node1->x0);
        newNode->v = 0.5f * (node0->v + node1->v);
        Edge* newEdges[2];
        newEdges[0] = new Edge(newNode, node0);
        newEdges[1] = new Edge(newNode, node1);
        
        addedNodes[i] = newNode;
        addedEdges[4 * i] = newEdges[0];
        addedEdges[4 * i + 1] = newEdges[1];
        removedEdges[i] = const_cast<Edge*>(edge);

        Vertex* newVertices[2];
        if (edge->isSeam()) {
            newVertices[0] = new Vertex(0.5f * (edge->vertices[0][0]->u + edge->vertices[0][1]->u));
            newVertices[0]->sizing = 0.5f * (edge->vertices[0][0]->sizing + edge->vertices[0][1]->sizing);
            newVertices[1] = new Vertex(0.5f * (edge->vertices[1][0]->u + edge->vertices[1][1]->u));
            newVertices[1]->sizing = 0.5f * (edge->vertices[1][0]->sizing + edge->vertices[1][1]->sizing);
            newVertices[0]->node = newVertices[1]->node = newNode;
            addedVertices[2 * i] = newVertices[0];
            addedVertices[2 * i + 1] = newVertices[1];
        } else {
            int j = edge->opposites[0] != nullptr ? 0 : 1;
            newVertices[0] = newVertices[1] = new Vertex(0.5f * (edge->vertices[j][0]->u + edge->vertices[j][1]->u));
            newVertices[0]->sizing = 0.5f * (edge->vertices[j][0]->sizing + edge->vertices[j][1]->sizing);
            newVertices[0]->node = newNode;
            addedVertices[2 * i] = newVertices[0];
            addedVertices[2 * i + 1] = nullptr;
        }

        for (int j = 0; j < 2; j++)
            if (edge->opposites[j] != nullptr) {
                Vertex* vertex0 = edge->vertices[j][j];
                Vertex* vertex1 = edge->vertices[j][1 - j];
                Vertex* vertex2 = edge->opposites[j];

                Face* face = edge->adjacents[j];
                Edge* edge0 = face->findEdge(vertex1, vertex2);
                Edge* edge1 = face->findEdge(vertex2, vertex0);
                
                Vertex* newVertex = newVertices[j];
                Edge* newEdge0 = newEdges[j];
                Edge* newEdge1 = newEdges[1 - j];
                Edge* newEdge2 = new Edge(newNode, vertex2->node);
                Face* newFace0 = new Face(vertex0, newVertex, vertex2, material);
                Face* newFace1 = new Face(vertex2, newVertex, vertex1, material);
                
                newEdge0->initialize(vertex2, newFace0);
                newEdge1->initialize(vertex2, newFace1);
                newEdge2->initialize(vertex0, newFace0);
                newEdge2->initialize(vertex1, newFace1);
                newFace0->setEdges(newEdge0, newEdge2, edge1);
                newFace1->setEdges(newEdge2, newEdge1, edge0);
                edge0->initialize(newVertex, newFace1);
                edge1->initialize(newVertex, newFace0);
                
                addedEdges[4 * i + j + 2] = newEdge2;
                addedFaces[4 * i + 2 * j] = newFace0;
                addedFaces[4 * i + 2 * j + 1] = newFace1;
                removedFaces[2 * i + j] = face;
            } else {
                addedEdges[4 * i + j + 2] = nullptr;
                addedFaces[4 * i + 2 * j] = addedFaces[4 * i + 2 * j + 1] = nullptr;
                removedFaces[2 * i + j] = nullptr;
            }
    }
}

__global__ void collectAdjacentEdges(int nEdges, const Edge* const* edges, int* indices, Edge** adjacentEdges) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        Edge* edge = const_cast<Edge*>(edges[i]);
        for (int j = 0; j < 2; j++) {
            int index = 2 * i + j;
            indices[index] = edge->nodes[j]->index;
            adjacentEdges[index] = edge;
        }
    }
}

__global__ void collectAdjacentFaces(int nFaces, const Face* const* faces, int* indices, Face** adjacentFaces) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nFaces; i += nThreads) {
        Face* face = const_cast<Face*>(faces[i]);
        for (int j = 0; j < 3; j++) {
            int index = 3 * i + j;
            indices[index] = face->vertices[j]->index;
            adjacentFaces[index] = face;
        }
    }
}

__global__ void setRange(int n, const int* indices, int* l, int* r) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += nThreads) {
        int index = indices[i];
        if (i == 0 || index != indices[i - 1])
            l[index] = i;
        if (i == n - 1 || index != indices[i + 1])
            r[index] = i + 1;
    }
}

__device__ bool shouldCollapseGpu(const Edge* edge, int side, const int* edgeBegin, const int* edgeEnd, const Edge* const* adjacentEdges, const int* faceBegin, const int* faceEnd, const Face* const* adjacentFaces, const Remeshing* remeshing) {
    Node* node = edge->nodes[side];
    if (node->preserve)
        return false;
    
    bool flag = false;
    int l = edgeBegin[node->index], r = edgeEnd[node->index];
    for (int i = l; i < r; i++) {
        const Edge* adjacentEdge = adjacentEdges[i];
        if (adjacentEdge->isBoundary() || adjacentEdge->isSeam()) {
            flag = true;
            break;
        }
    }
    if (flag && (!edge->isBoundary() && !edge->isSeam()))
        return false;
    
    if (edge->isSeam())
        for (int i = 0; i < 2; i++) {
            Vertex* vertex0 = edge->vertices[i][side];
            Vertex* vertex1 = edge->vertices[i][1 - side];
            
            int l = faceBegin[vertex0->index], r = faceEnd[vertex0->index];
            for (int j = l; j < r; j++) {
                const Face* adjacentFace = adjacentFaces[j];
                Vertex* v0 = adjacentFace->vertices[0];
                Vertex* v1 = adjacentFace->vertices[1];
                Vertex* v2 = adjacentFace->vertices[2];
                if (v0 == vertex1 || v1 == vertex1 || v2 == vertex1)
                    continue;
                
                if (v0 == vertex0)
                    v0 = vertex1;
                else if (v1 == vertex0) {
                    v1 = vertex1;
                    mySwap(v0, v1);
                } else {
                    v2 = vertex1;
                    mySwap(v0, v2);
                }
                Vector2f u0 = v0->u;
                Vector2f u1 = v1->u;
                Vector2f u2 = v2->u;
                float a = 0.5f * (u1 - u0).cross(u2 - u0);
                float p = (u0 - u1).norm() + (u1 - u2).norm() + (u2 - u0).norm();
                float aspect = 12.0f * sqrt(3.0f) * a / sqr(p);
                if (a < 1e-6f || aspect < remeshing->aspectMin)
                    return false;
                if (edgeMetric(v0, v1) > 0.9f || edgeMetric(v0, v2) > 0.9f)
                    return false;
            }
        }
    else {
        int index = edge->opposites[0] != nullptr ? 0 : 1;
        Vertex* vertex0 = edge->vertices[index][side];
        Vertex* vertex1 = edge->vertices[index][1 - side];

        int l = faceBegin[vertex0->index], r = faceEnd[vertex0->index];
        for (int i = l; i < r; i++) {
            const Face* adjacentFace = adjacentFaces[i];
            Vertex* v0 = adjacentFace->vertices[0];
            Vertex* v1 = adjacentFace->vertices[1];
            Vertex* v2 = adjacentFace->vertices[2];
            if (v0 == vertex1 || v1 == vertex1 || v2 == vertex1)
                continue;
            
            if (v0 == vertex0)
                v0 = vertex1;
            else if (v1 == vertex0) {
                v1 = vertex1;
                mySwap(v0, v1);
            } else {
                v2 = vertex1;
                mySwap(v0, v2);
            }
            Vector2f u0 = v0->u;
            Vector2f u1 = v1->u;
            Vector2f u2 = v2->u;
            float a = 0.5f * (u1 - u0).cross(u2 - u0);
            float p = (u0 - u1).norm() + (u1 - u2).norm() + (u2 - u0).norm();
            float aspect = 12.0f * sqrt(3.0f) * a / sqr(p);
            if (a < 1e-6f || aspect < remeshing->aspectMin)
                return false;
            if (edgeMetric(v0, v1) > 0.9f || edgeMetric(v0, v2) > 0.9f)
                return false;
        }
    }

    return true;
}

__global__ void checkEdgesToCollapse(int nEdges, const Edge* const* edges, const int* edgeBegin, const int* edgeEnd, const Edge* const* adjacentEdges, const int* faceBegin, const int* faceEnd, const Face* const* adjacentFaces, const Remeshing* remeshing, Pairei* edgesToCollapse) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        const Edge* edge = edges[i];
        if (shouldCollapseGpu(edge, 0, edgeBegin, edgeEnd, adjacentEdges, faceBegin, faceEnd, adjacentFaces, remeshing))
            edgesToCollapse[i] = Pairei(const_cast<Edge*>(edge), 0);
        else if (shouldCollapseGpu(edge, 1, edgeBegin, edgeEnd, adjacentEdges, faceBegin, faceEnd, adjacentFaces, remeshing))
            edgesToCollapse[i] = Pairei(const_cast<Edge*>(edge), 1);
        else
            edgesToCollapse[i] = Pairei(nullptr, -1);
    }
}

__global__ void initializeCollapseNodes(int nEdges, const Pairei* edges, const int* edgeBegin, const int* edgeEnd, const Edge* const* adjacentEdges) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        Edge* edge = edges[i].first;
        int side = edges[i].second;
        
        Node* node = edge->nodes[side];
        node->removed = false;
        int l = edgeBegin[node->index], r = edgeEnd[node->index];
        for (int j = l; j < r; j++) {
            const Edge* adjacentEdge = adjacentEdges[j];
            Node* adjacentNode = adjacentEdge->nodes[0] != node ? adjacentEdge->nodes[0] : adjacentEdge->nodes[1];
            adjacentNode->removed = false;
        }
    }
}

__global__ void resetCollapseNodes(int nEdges, const Pairei* edges, const int* edgeBegin, const int* edgeEnd, const Edge* const* adjacentEdges) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        Edge* edge = edges[i].first;
        int side = edges[i].second;

        Node* node = edge->nodes[side];
        if (!node->removed) {
            int l = edgeBegin[node->index], r = edgeEnd[node->index];
            bool flag = true;
            for (int j = l; j < r; j++) {
                const Edge* adjacentEdge = adjacentEdges[j];
                Node* adjacentNode = adjacentEdge->nodes[0] != node ? adjacentEdge->nodes[0] : adjacentEdge->nodes[1];
                if (adjacentNode->removed) {
                    flag = false;
                    break;
                }
            }

            if (flag) {
                node->minIndex = nEdges;
                for (int j = l; j < r; j++) {
                    const Edge* adjacentEdge = adjacentEdges[j];
                    Node* adjacentNode = adjacentEdge->nodes[0] != node ? adjacentEdge->nodes[0] : adjacentEdge->nodes[1];
                    adjacentNode->minIndex = nEdges;
                }
            }
        }
    }
}

__global__ void computeCollapseMinIndices(int nEdges, const Pairei* edges, const int* edgeBegin, const int* edgeEnd, const Edge* const* adjacentEdges) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        Edge* edge = edges[i].first;
        int side = edges[i].second;

        Node* node = edge->nodes[side];
        if (!node->removed) {
            int l = edgeBegin[node->index], r = edgeEnd[node->index];
            bool flag = true;
            for (int j = l; j < r; j++) {
                const Edge* adjacentEdge = adjacentEdges[j];
                Node* adjacentNode = adjacentEdge->nodes[0] != node ? adjacentEdge->nodes[0] : adjacentEdge->nodes[1];
                if (adjacentNode->removed) {
                    flag = false;
                    break;
                }
            }

            if (flag) {
                atomicMin(&node->minIndex, i);
                for (int j = l; j < r; j++) {
                    const Edge* adjacentEdge = adjacentEdges[j];
                    Node* adjacentNode = adjacentEdge->nodes[0] != node ? adjacentEdge->nodes[0] : adjacentEdge->nodes[1];
                    atomicMin(&adjacentNode->minIndex, i);
                }
            }
        }
    }
}

__global__ void checkIndependentEdgesToCollapse(int nEdges, const Pairei* edges, const int* edgeBegin, const int* edgeEnd, const Edge* const* adjacentEdges, Pairei* edgesToCollapse) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        Edge* edge = edges[i].first;
        int side = edges[i].second;

        Node* node = edge->nodes[side];
        if (!node->removed && node->minIndex == i) {
            int l = edgeBegin[node->index], r = edgeEnd[node->index];
            bool flag = true;
            for (int j = l; j < r; j++) {
                const Edge* adjacentEdge = adjacentEdges[j];
                Node* adjacentNode = adjacentEdge->nodes[0] != node ? adjacentEdge->nodes[0] : adjacentEdge->nodes[1];
                if (adjacentNode->removed || adjacentNode->minIndex != i) {
                    flag = false;
                    break;
                }
            }

            if (flag) {
                edgesToCollapse[i] = edges[i];
                node->removed = true;
                for (int j = l; j < r; j++) {
                    const Edge* adjacentEdge = adjacentEdges[j];
                    Node* adjacentNode = adjacentEdge->nodes[0] != node ? adjacentEdge->nodes[0] : adjacentEdge->nodes[1];
                    adjacentNode->removed = true;
                }
            }
        }
    }
}

__global__ void collapseGpu(int nEdges, const Pairei* edges, const Material* material, const int* edgeBegin, const int* edgeEnd, Edge* const* adjacentEdges, const int* faceBegin, const int* faceEnd, Face* const* adjacentFaces, Node** removedNodes, Vertex** removedVertices, Edge** removedEdges, Face** removedFaces) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        Edge* edge = edges[i].first;
        int side = edges[i].second;
        Node* node0 = edge->nodes[side];
        Node* node1 = edge->nodes[1 - side];

        removedNodes[i] = node0;
        removedEdges[3 * i] = edge;

        for (int j = 0; j < 2; j++)
            if (edge->opposites[j] != nullptr) {
                Vertex* vertex0 = edge->vertices[j][side];
                Vertex* vertex1 = edge->vertices[j][1 - side];
                Vertex* vertex2 = edge->opposites[j];

                Face* face = edge->adjacents[j];
                Edge* edge0 = face->findEdge(vertex1, vertex2);
                Edge* edge1 = face->findEdge(vertex2, vertex0);

                if (!edge1->isBoundary()) {
                    Vertex* v = edge1->opposites[0];
                    Face* f = edge1->adjacents[0];
                    if (f == face) {
                        v = edge1->opposites[1];
                        f = edge1->adjacents[1];
                    }

                    edge0->replaceOpposite(vertex0, v);
                    edge0->replaceAdjacent(face, f);
                    f->replaceEdge(edge1, edge0);
                }

                removedEdges[3 * i + j + 1] = edge1;
                removedFaces[2 * i + j] = face;
            } else {
                removedEdges[3 * i + j + 1] = nullptr;
                removedFaces[2 * i + j] = nullptr;
            }

        int l = edgeBegin[node0->index], r = edgeEnd[node0->index];
        for (int j = l; j < r; j++) {
            Edge* adjacentEdge = adjacentEdges[j];
            if (adjacentEdge != edge) {
                adjacentEdge->replaceNode(node0, node1);
                adjacentEdge->replaceVertex(edge->vertices[0][side], edge->vertices[0][1 - side]);
                adjacentEdge->replaceVertex(edge->vertices[1][side], edge->vertices[1][1 - side]);
            }
        }

        if (edge->isSeam())
            for (int j = 0; j < 2; j++) {
                Vertex* vertex0 = edge->vertices[j][side];
                Vertex* vertex1 = edge->vertices[j][1 - side];
                removedVertices[2 * i + j] = vertex0;

                l = faceBegin[vertex0->index];
                r = faceEnd[vertex0->index];
                for (int k = l; k < r; k++) {
                    Face* adjacentFace = adjacentFaces[k];
                    if (adjacentFace != edge->adjacents[0] && adjacentFace != edge->adjacents[1]) {
                        adjacentFace->findOpposite(vertex0)->replaceOpposite(vertex0, vertex1);
                        adjacentFace->replaceVertex(vertex0, vertex1);
                        adjacentFace->initialize(material);
                    }
                }
            }
        else {
            int index = edge->opposites[0] != nullptr ? 0 : 1;
            Vertex* vertex0 = edge->vertices[index][side];
            Vertex* vertex1 = edge->vertices[index][1 - side];
            removedVertices[2 * i] = vertex0;
            removedVertices[2 * i + 1] = nullptr;
            
            l = faceBegin[vertex0->index];
            r = faceEnd[vertex0->index];
            for (int k = l; k < r; k++) {
                Face* adjacentFace = adjacentFaces[k];
                if (adjacentFace != edge->adjacents[0] && adjacentFace != edge->adjacents[1]) {
                    adjacentFace->findOpposite(vertex0)->replaceOpposite(vertex0, vertex1);
                    adjacentFace->replaceVertex(vertex0, vertex1);
                    adjacentFace->initialize(material);
                }
            }
        }
    }
}

__global__ void printEdges(int nEdges, const Pairei* edges) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        Edge* edge = edges[i].first;
        int side = edges[i].second;
        printf("%d %d %d\n", edge->nodes[0]->index, edge->nodes[1]->index, side);
    }
}
#include "hip/hip_runtime.h"
#include "MeshHelper.cuh"

__global__ void initializeNodes(int nNodes, const Vector3f* x, bool isFree, Node** nodes) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads)
        nodes[i] = new Node(x[i], isFree);
}

__global__ void initializeVertices(int nVertices, const Vector2f* u, Vertex** vertices) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nVertices; i += nThreads)
        vertices[i] = new Vertex(u[i]);
}

__device__ void setEdgeData(int index0, int index1, const Vertex* vertex, const Face* face, Pairii& index, EdgeData& edgeData) {
    if (index0 > index1)
        mySwap(index0, index1);
    
    index.first = index0;
    index.second = index1;
    edgeData.opposite = const_cast<Vertex*>(vertex);
    edgeData.adjacent = const_cast<Face*>(face);
}

__global__ void initializeFaces(int nFaces, const int* xIndices, const int* uIndices, const Node* const* nodes, const Material* material, Vertex** vertices, Face** faces, Pairii* edgeIndices, EdgeData* edgeData) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nFaces; i += nThreads) {
        int index0 = 3 * i;
        int index1 = 3 * i + 1;
        int index2 = 3 * i + 2;
        int xIndex0 = xIndices[index0];
        int xIndex1 = xIndices[index1];
        int xIndex2 = xIndices[index2];
        int uIndex0 = uIndices[index0];
        int uIndex1 = uIndices[index1];
        int uIndex2 = uIndices[index2];
        const Node* node0 = nodes[xIndex0];
        const Node* node1 = nodes[xIndex1];
        const Node* node2 = nodes[xIndex2];
        Vertex* vertex0 = vertices[uIndex0];
        Vertex* vertex1 = vertices[uIndex1];
        Vertex* vertex2 = vertices[uIndex2];
        vertex0->node = const_cast<Node*>(node0);
        vertex1->node = const_cast<Node*>(node1);
        vertex2->node = const_cast<Node*>(node2);
        Face* face = new Face(vertex0, vertex1, vertex2, material);
        setEdgeData(xIndex0, xIndex1, vertex2, face, edgeIndices[index0], edgeData[index0]);
        setEdgeData(xIndex1, xIndex2, vertex0, face, edgeIndices[index1], edgeData[index1]);
        setEdgeData(xIndex2, xIndex0, vertex1, face, edgeIndices[index2], edgeData[index2]);
        faces[i] = face;
    }
}

__global__ void initializeEdges(int nEdges, const Pairii* indices, const EdgeData* edgeData, const Node* const* nodes, Edge** edges) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads)
        if (i == 0 || indices[i] != indices[i - 1]) {
            const Pairii& index = indices[i];
            const EdgeData& e = edgeData[i];
            Edge* edge = new Edge(nodes[index.first], nodes[index.second]);
            edge->initialize(e.opposite, e.adjacent);
            e.adjacent->setEdge(edge);
            edges[i] = edge;
        } else
            edges[i] = nullptr;
}

__global__ void setEdges(int nEdges, const Pairii* indices, const EdgeData* edgeData, Edge** edges) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads)
        if (i > 0 && indices[i] == indices[i - 1]) {
            const EdgeData& e = edgeData[i];
            Edge* edge = edges[i - 1];
            edge->initialize(e.opposite, e.adjacent);
            e.adjacent->setEdge(edge);
        }
}

__global__ void setPreserve(int nEdges, const Edge* const* edges) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        const Edge* edge = edges[i];
        if (edge->isBoundary() || edge->isSeam())
            for (int j = 0; j < 2; j++)
                edge->nodes[j]->preserve = true;
    }
}

__global__ void resetGpu(int nNodes, Node** nodes) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads) {
        Node* node = nodes[i];
        node->x = node->x0;
    }
}

__global__ void setBackupFaces(int nFaces, const Face* const* faces, BackupFace* backupFaces) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nFaces; i += nThreads) {
        BackupFace& backupFace = backupFaces[i];
        const Face* face = faces[i];
        for (int j = 0; j < 3; j++) {
            Vertex* vertex = face->vertices[j];
            backupFace.x[j] = vertex->node->x;
            backupFace.u[j] = vertex->u;
        }
    }
}

__global__ void initializeIndices(int n, int* indices) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += nThreads)
        indices[i] = i;
}

__global__ void initializeNodeStructures(int nNodes, Node** nodes) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads) {
        Node* node = nodes[i];
        node->index = i;
        node->mass = 0.0f;
        node->area = 0.0f;
    }
}

__global__ void initializeVertexStructures(int nVertices, Vertex** vertices) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nVertices; i += nThreads)
        vertices[i]->index = i;
}

__global__ void updateStructuresGpu(int nFaces, const Face* const* faces) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nFaces; i += nThreads) {
        const Face* face = faces[i];
        float mass = face->mass / 3.0f;
        float area = face->area;
        for (int j = 0; j < 3; j++) {
            Node* node = face->vertices[j]->node;
            atomicAdd(&node->mass, mass);
            atomicAdd(&node->area, area);
        }
    }
}

__global__ void initializeNodeGeometries(int nNodes, Node** nodes) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads) {
        Node* node = nodes[i];
        node->x1 = node->x;
        node->n = Vector3f();
    }
}

__global__ void updateNodeGeometriesGpu(int nFaces, const Face* const* faces) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nFaces; i += nThreads) {
        const Face* face = faces[i];
        for (int j = 0; j < 3; j++) {
            Node* node = face->vertices[j]->node;
            Vector3f e0 = face->vertices[(j + 1) % 3]->node->x - node->x;
            Vector3f e1 = face->vertices[(j + 2) % 3]->node->x - node->x;
            Vector3f n = e0.cross(e1) / (e0.norm2() * e1.norm2());
            for (int k = 0; k < 3; k++)
                atomicAdd(&node->n(k), n(k));
        }
    }
}

__global__ void finalizeNodeGeometries(int nNodes, Node** nodes) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads)
        nodes[i]->n.normalize();
}

__global__ void updateFaceGeometriesGpu(int nFaces, Face** faces) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nFaces; i += nThreads)
        faces[i]->update();
}

__global__ void updateVelocitiesGpu(int nNodes, float invDt, Node** nodes) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads) {
        Node* node = nodes[i];
        node->v = (node->x - node->x0) * invDt;
    }
}

__global__ void updateRenderingDataGpu(int nFaces, const Face* const* faces, Renderable* renderables) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nFaces; i += nThreads) {
        const Face* face = faces[i];
        for (int j = 0; j < 3; j++) {
            Vertex* vertex = face->vertices[j];
            Node* node = vertex->node;
            int index = 3 * i + j;
            renderables[index].x = node->x;
            renderables[index].n = node->n;
            renderables[index].u = vertex->u;
        }
    }
}

__global__ void copyX(int nNodes, const Node* const* nodes, Vector3f* x) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads)
        x[i] = nodes[i]->x;
}

__global__ void copyV(int nNodes, const Node* const* nodes, Vector3f* v) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads)
        v[i] = nodes[i]->v;
}

__global__ void copyU(int nVertices, const Vertex* const* vertices, Vector2f* u) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nVertices; i += nThreads)
        u[i] = vertices[i]->u;
}

__global__ void copyFaceIndices(int nFaces, const Face* const* faces, Pairii* indices) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nFaces; i += nThreads) {
        const Face* face = faces[i];
        for (int j = 0; j < 3; j++) {
            const Vertex* vertex = face->vertices[j];
            Pairii& index = indices[3 * i + j];
            index.first = vertex->node->index;
            index.second = vertex->index;
        }
    }
}

__global__ void printDebugInfoGpu(const Face* const* faces, int index) {
    const Face* face = faces[index];
    printf("Nodes=[%d, %d, %d]\n", face->vertices[0]->node->index, face->vertices[1]->node->index, face->vertices[2]->node->index);
}

__global__ void checkEdges(int nEdges, const Edge* const* edges) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nEdges; i += nThreads) {
        const Edge* edge = edges[i];
        for (int j = 0; j < 2; j++)
            if (edge->opposites[j] != nullptr) {
                if (edge->vertices[j][0]->node != edge->nodes[0] || edge->vertices[j][1]->node != edge->nodes[1])
                    printf("Edge vertices check error!\n");
                if (edge->adjacents[j] == nullptr || !edge->adjacents[j]->contain(edge->opposites[j]) || !edge->adjacents[j]->contain(edge))
                    printf("Edge adjacents check error!\n");
            } else if (edge->adjacents[j] != nullptr)
                printf("Edge opposites check error!\n");
    }
}

__global__ void checkFaces(int nFaces, const Face* const* faces) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nFaces; i += nThreads) {
        const Face* face = faces[i];
        for (int j = 0; j < 3; j++) {
            Edge* edge = face->edges[j];
            if (edge->adjacents[0] != face && edge->adjacents[1] != face)
                printf("Face edges check error!\n");
        }
    }
}
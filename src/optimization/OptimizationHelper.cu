#include "hip/hip_runtime.h"
#include "OptimizationHelper.cuh"

float clampViolation(float x, int sign) {
    return sign < 0 ? max(x, 0.0f) : (sign > 0 ? min(x, 0.0f) : x);
}

__global__ void setDiff(int nNodes, const Node* const* nodes, int* diff) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads)
        diff[i] = (i > 0 && nodes[i] != nodes[i - 1]);
}

__global__ void setIndices(int nNodes, const int* nodeIndices, const int* diff, int* indices) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads)
        indices[nodeIndices[i]] = diff[i];
}

__global__ void initializeGpu(int nNodes, const Node* const* nodes, Vector3f* x) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads)
        x[i] = nodes[i]->x;
}

__global__ void finalizeGpu(int nNodes, const Vector3f* x, Node** nodes) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads)
        nodes[i]->x = x[i];
}

__global__ void collectCollisionNodes(int nConstraints, const Impact* impacts, int deform, int* indices, Node** nodes) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nConstraints; i += nThreads) {
        const Impact& impact = impacts[i];
        for (int j = 0; j < 4; j++) {
            int index = 4 * i + j;
            Node* node = impact.nodes[j];
            if (deform == 1 || node->isFree) {
                indices[index] = index;
                nodes[index] = node;
            } else {
                indices[index] = -1;
                nodes[index] = nullptr;
            }
        }
    }
}

__global__ void collisionInv(int nNodes, const Node* const* nodes, float obstacleMass, float* inv) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads) {
        const Node* node = nodes[i];
        float mass = node->isFree ? node->mass : obstacleMass;
        inv[i] = 1.0f / mass;
    }
}

__global__ void collisionObjective(int nNodes, const Node* const* nodes, float obstacleMass, const Vector3f* x, float* objectives) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads) {
        const Node* node = nodes[i];
        float mass = node->isFree ? node->mass : obstacleMass;
        objectives[i] = mass * (x[i] - node->x1).norm2();
    }
}

__global__ void collisionObjectiveGradient(int nNodes, const Node* const* nodes, float invMass, float obstacleMass, const Vector3f* x, Vector3f* gradient) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads) {
        const Node* node = nodes[i];
        float mass = node->isFree ? node->mass : obstacleMass;
        gradient[i] = invMass * mass * (x[i] - node->x1);
    }
}

__global__ void collisionConstraint(int nConstraints, const Impact* impacts, const int* indices, float thickness, const Vector3f* x, float* constraints, int* signs) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nConstraints; i += nThreads) {
        float c = -thickness;
        const Impact& impact = impacts[i];
        for (int j = 0; j < 4; j++) {
            int k = indices[4 * i + j];
            if (k > -1)
                c += impact.w[j] * impact.n.dot(x[k]);
            else
                c += impact.w[j] * impact.n.dot(impact.nodes[j]->x);
        }
        constraints[i] = c;
        signs[i] = 1;
    }
}

__global__ void collectCollisionConstraintGradient(int nConstraints, const Impact* impacts, const float* coefficients, float mu, Vector3f* grad) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nConstraints; i += nThreads) {
        const Impact& impact = impacts[i];
        for (int j = 0; j < 4; j++)
            grad[4 * i + j] = mu * coefficients[i] * impact.w[j] * impact.n;
    }
}

__global__ void collectSeparationNodes(int nConstraints, const Intersection* intersections, int deform, int* indices, Node** nodes) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nConstraints; i += nThreads) {
        const Intersection& intersection = intersections[i];
        Face* face0 = intersection.face0;
        Face* face1 = intersection.face1;

        for (int j = 0; j < 3; j++) {
            int index0 = 6 * i + j;
            Node* node0 = face0->vertices[j]->node;
            if (deform == 1 || node0->isFree) {
                indices[index0] = index0;
                nodes[index0] = node0;
            } else {
                indices[index0] = -1;
                nodes[index0] = nullptr;
            }

            int index1 = 6 * i + j + 3;
            Node* node1 = face1->vertices[j]->node;
            if (deform == 1 || node1->isFree) {
                indices[index1] = index1;
                nodes[index1] = node1;
            } else {
                indices[index1] = -1;
                nodes[index1] = nullptr;
            }
        }
    }
}

__global__ void separationInv(int nNodes, const Node* const* nodes, float obstacleArea, float* inv) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads) {
        const Node* node = nodes[i];
        float area = node->isFree ? node->area : obstacleArea;
        inv[i] = 1.0f / area;
    }
}

__global__ void separationObjective(int nNodes, const Node* const* nodes, float obstacleArea, const Vector3f* x, float* objectives) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads) {
        const Node* node = nodes[i];
        float area = node->isFree ? node->area : obstacleArea;
        objectives[i] = area * (x[i] - node->x1).norm2();
    }
}

__global__ void separationObjectiveGradient(int nNodes, const Node* const* nodes, float invArea, float obstacleArea, const Vector3f* x, Vector3f* gradient) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads) {
        const Node* node = nodes[i];
        float area = node->isFree ? node->area : obstacleArea;
        gradient[i] = invArea * area * (x[i] - node->x1);
    }
}

__global__ void separationConstraint(int nConstraints, const Intersection* intersections, const int* indices, float thickness, const Vector3f* x, float* constraints, int* signs) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nConstraints; i += nThreads) {
        float c = -thickness;
        const Intersection& intersection = intersections[i];
        for (int j = 0; j < 3; j++) {
            int k0 = indices[6 * i + j];
            if (k0 > -1)
                c += intersection.b0(j) * intersection.d.dot(x[k0]);
            else
                c += intersection.b0(j) * intersection.d.dot(intersection.face0->vertices[j]->node->x);

            int k1 = indices[6 * i + j + 3];
            if (k1 > -1)
                c -= intersection.b1(j) * intersection.d.dot(x[k1]);
            else
                c -= intersection.b1(j) * intersection.d.dot(intersection.face1->vertices[j]->node->x);
        }
        constraints[i] = c;
        signs[i] = 1;
    }
}

__global__ void collectSeparationConstraintGradient(int nConstraints, const Intersection* intersections, const float* coefficients, float mu, Vector3f* grad) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nConstraints; i += nThreads) {
        const Intersection& intersection = intersections[i];
        for (int j = 0; j < 3; j++) {
            grad[6 * i + j] = mu * coefficients[i] * intersection.b0(j) * intersection.d;
            grad[6 * i + j + 3] = -mu * coefficients[i] * intersection.b1(j) * intersection.d;
        }
    }
}

__global__ void addConstraintGradient(int nIndices, const int* indices, const Vector3f* grad, Vector3f* gradtient) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nIndices; i += nThreads)
        gradtient[indices[i]] += grad[i];
}

__global__ void computeCoefficient(int nConstraints, const float* lambda, float mu, const int* signs, float* c) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nConstraints; i += nThreads)
        c[i] = clampViolation(c[i] + lambda[i] / mu, signs[i]);
}

__global__ void computeSquare(int nConstraints, float* c) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nConstraints; i += nThreads)
        c[i] = sqr(c[i]);
}

__global__ void computeNorm2(int nNodes, const Vector3f* x, float* x2) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads)
        x2[i] = x[i].norm2();
}

__global__ void computeXt(int nNodes, const Vector3f* x, const Vector3f* gradient, float s, Vector3f* xt) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads)
        xt[i] = x[i] - s * gradient[i];
}

__global__ void updateX(int nNodes, const Vector3f* gradient, float s, Vector3f* x) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nNodes; i += nThreads)
        x[i] -= s * gradient[i];
}

__global__ void updateMultiplierGpu(int nConstraints, const float* c, const int* signs, float mu, float* lambda) {
    int nThreads = gridDim.x * blockDim.x;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < nConstraints; i += nThreads)
        lambda[i] = clampViolation(lambda[i] + mu * c[i], signs[i]);
}
